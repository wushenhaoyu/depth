#include "hip/hip_runtime.h"
#include "DataParameter.cuh"
using namespace cv;

using namespace std;



__constant__ RawImageParameter d_rawImageParameter;
__constant__ DisparityParameter d_disparityParameter;
__constant__ FilterParameterDevice d_filterPatameterDevice; 
__device__ MicroImageParameterDevice d_microImageParameter; 
__device__ float* d_costVol;
__device__ float* d_rawDisp;
__device__ float* d_ppLensMeanDisp;
__device__ float* d_renderCache;
__device__ float* d_inputImg;
__device__ float* d_inputImgRec;
__device__ RanderMapPatch* d_ppRanderMapPatch;
__device__ float* d_tmp;
__device__ float* d_simg;
__device__ int *d_sx_begin, *d_sy_begin, *d_sx_end, *d_sy_end;
__device__ int *d_randerMapWidth, *d_randerMapHeight;

__constant__ float d_fltMax;
__constant__ int d_meanDispLenRadius;
__constant__ int d_patchScale9;
__constant__ float d_randerScale;
__constant__ int d_destWidth;
__constant__ int d_destHeight;

__global__ void testKernel() {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // 在一个特定的索引位置修改 d_ppLensMeanDisp
   // printf("m_xLensNum = %d, m_yLensNum = %d\n", d_rawImageParameter.m_xLensNum, d_rawImageParameter.m_yLensNum);
    
    
    if (x < d_rawImageParameter.m_xLensNum && y < d_rawImageParameter.m_yLensNum) {
        int index = y * d_rawImageParameter.m_xLensNum + x;
        if (index == 0) {  
			if (d_ppLensMeanDisp == nullptr) {
				printf("d_ppLensMeanDisp is NULL\n");
			}
			else{
				printf("d_ppLensMeanDisp is not NULL\n");
			}
            d_ppLensMeanDisp[index] = 0;  
        }
    }
}

__global__ void testRanderMapPatchKernel(RanderMapPatch* d_ppRanderMapPatch, int numPatches, int width, int height)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPatches) return;

    // 获取当前 RanderMapPatch 的 simg 指针
    float* simg = d_ppRanderMapPatch[idx].simg;

    // 对 simg 指向的内存进行简单的写操作
    for (int i = 0; i < width * height * 3; ++i)
    {
        simg[i] = static_cast<float>(idx * (i + 1)); // 写入一些模式数据
    }

    // 设置 sy 和 sx 的值
    d_ppRanderMapPatch[idx].sy = idx * 10; // 示例值
    d_ppRanderMapPatch[idx].sx = idx * 20; // 示例值


    printf("Patch 0: sy = %d, sx = %d, simg[0] = %f, simg[1] = %f, simg[2] = %f\n",
               d_ppRanderMapPatch[idx].sy, d_ppRanderMapPatch[idx].sx,
               simg[0], simg[1], simg[2]);
    
}

__global__ void testAssignAndPrintKernel()
{
    // 每个线程对全局变量赋值
    *d_sx_begin = 10;
    *d_sy_begin = 20;
    *d_sx_end = 30;
    *d_sy_end = 40;

    // 打印全局变量的值
    printf("d_sx_begin: %d\n", *d_sx_begin);
    printf("d_sy_begin: %d\n", *d_sy_begin);
    printf("d_sx_end: %d\n", *d_sx_end);
    printf("d_sy_end: %d\n", *d_sy_end);
}

void DataParameter::mapToGPU() {
    // 复制常量变量
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_rawImageParameter), &m_rawImageParameter, sizeof(RawImageParameter)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_disparityParameter), &m_disparityParameter, sizeof(DisparityParameter)));

    float fltMax = FLT_MAX;
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_fltMax), &fltMax, sizeof(float)));

    int meanDispLenRadius = MEAN_DISP_LEN_RADIUS;
    int patchScale9 = PATCH_SCALE9;
    float randerScale = RANDER_SCALE;
    int destWidth = DEST_WIDTH;
    int destHeight = DEST_HEIGHT;

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_meanDispLenRadius), &meanDispLenRadius, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_patchScale9), &patchScale9, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_randerScale), &randerScale, sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_destWidth), &destWidth, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_destHeight), &destHeight, sizeof(int)));

    // 在主机端分配内存并将指针传递给设备端
    float* h_costVol;
    CUDA_CHECK(hipMalloc((void**)&h_costVol, m_disparityParameter.m_disNum * m_rawImageParameter.m_recImgHeight * m_rawImageParameter.m_recImgWidth * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_costVol), &h_costVol, sizeof(float*)));

    float* h_rawDisp;
    CUDA_CHECK(hipMalloc((void**)&h_rawDisp, m_rawImageParameter.m_recImgHeight * m_rawImageParameter.m_recImgWidth * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_rawDisp), &h_rawDisp, sizeof(float*)));
	

    float* h_ppLensMeanDisp;
    CUDA_CHECK(hipMalloc((void**)&h_ppLensMeanDisp, m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_ppLensMeanDisp), &h_ppLensMeanDisp, sizeof(float*)));

    float* h_renderCache;
    CUDA_CHECK(hipMalloc((void**)&h_renderCache, m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_renderCache), &h_renderCache, sizeof(float*)));

    /*float* h_inputImg;
    CUDA_CHECK(hipMalloc((void**)&h_inputImg, m_rawImageParameter.m_srcImgHeight * m_rawImageParameter.m_srcImgWidth * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_inputImg), &h_inputImg, sizeof(float*)));

    float* h_inputImgRec;
    CUDA_CHECK(hipMalloc((void**)&h_inputImgRec, m_rawImageParameter.m_recImgHeight * m_rawImageParameter.m_recImgWidth * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_inputImgRec), &h_inputImgRec, sizeof(float*)));*/


int height = m_rawImageParameter.m_recImgHeight;
int width = m_rawImageParameter.m_recImgWidth;
int sizeOfFloat = (int)sizeof(float);

    float* h_tmp;
    CUDA_CHECK(hipMalloc((void**)&h_tmp, DEST_WIDTH * DEST_HEIGHT * 3 * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_tmp), &h_tmp, sizeof(float*)));

    float* h_simg;
    CUDA_CHECK(hipMalloc((void**)&h_simg, DEST_WIDTH * DEST_HEIGHT * 3 * sizeof(float)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_simg), &h_simg, sizeof(float*)));

	int* h_sx_begin;
	CUDA_CHECK(hipMalloc((void**)&h_sx_begin, sizeof(int)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sx_begin), &h_sx_begin, sizeof(int*)));


	int* h_sy_begin;
	CUDA_CHECK(hipMalloc((void**)&h_sy_begin, sizeof(int)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sy_begin), &h_sy_begin, sizeof(int*)));

	int* h_sx_end;
	CUDA_CHECK(hipMalloc((void**)&h_sx_end, sizeof(int)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sx_end), &h_sx_end, sizeof(int*)));

	int* h_sy_end;
	CUDA_CHECK(hipMalloc((void**)&h_sy_end, sizeof(int)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sy_end), &h_sy_end, sizeof(int*)));

	int* h_randerMapWidth;
    CUDA_CHECK(hipMalloc((void**)&h_randerMapWidth, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_randerMapWidth), &h_randerMapWidth, sizeof(int*)));

	int* h_randerMapHeight;
    CUDA_CHECK(hipMalloc((void**)&h_randerMapHeight, sizeof(int)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_randerMapHeight), &h_randerMapHeight, sizeof(int*)));

    // 为 MicroImageParameterDevice 分配内存并传递数据到设备
    MicroImageParameterDevice h_microImageParameterDevice;
    h_microImageParameterDevice.m_circleDiameter = m_microImageParameter.m_circleDiameter;
    h_microImageParameterDevice.m_circleNarrow = m_microImageParameter.m_circleNarrow;
    h_microImageParameterDevice.m_radiusDisEqu = m_microImageParameter.m_radiusDisEqu;

    int lensCenterPointsSize = m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * sizeof(cv::Point2d);
    CUDA_CHECK(hipMalloc((void**)&h_microImageParameterDevice.m_ppLensCenterPoints, lensCenterPointsSize));
    CUDA_CHECK(hipMemcpy(h_microImageParameterDevice.m_ppLensCenterPoints, m_microImageParameter.m_ppLensCenterPoints[0], lensCenterPointsSize, hipMemcpyHostToDevice));

    int pixelsMappingSetSize = m_rawImageParameter.m_srcImgHeight * m_rawImageParameter.m_srcImgWidth * sizeof(int);
    CUDA_CHECK(hipMalloc((void**)&h_microImageParameterDevice.m_ppPixelsMappingSet, pixelsMappingSetSize));
    CUDA_CHECK(hipMemcpy(h_microImageParameterDevice.m_ppPixelsMappingSet, m_microImageParameter.m_ppPixelsMappingSet[0], pixelsMappingSetSize, hipMemcpyHostToDevice));

    int matchNeighborLensSize = m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * NEIGHBOR_MATCH_LENS_NUM * sizeof(MatchNeighborLens);
    CUDA_CHECK(hipMalloc((void**)&h_microImageParameterDevice.m_ppMatchNeighborLens, matchNeighborLensSize));
    CUDA_CHECK(hipMemcpy(h_microImageParameterDevice.m_ppMatchNeighborLens, m_microImageParameter.m_ppMatchNeighborLens[0][0], matchNeighborLensSize, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_microImageParameter), &h_microImageParameterDevice, sizeof(MicroImageParameterDevice)));

    // 为 FilterParameterDevice 分配内存并传递数据到设备
    int* d_validNeighborPixelsNum;
    int* d_validPixelsMask;
    float* d_filterKernel;

    CUDA_CHECK(hipMalloc((void**)&d_validNeighborPixelsNum, m_filterPatameter.m_pValidNeighborPixelsNum->total() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_validNeighborPixelsNum, m_filterPatameter.m_pValidNeighborPixelsNum->data, 
                          m_filterPatameter.m_pValidNeighborPixelsNum->total() * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&d_validPixelsMask, m_filterPatameter.m_pValidPixelsMask->total() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_validPixelsMask, m_filterPatameter.m_pValidPixelsMask->data, 
                          m_filterPatameter.m_pValidPixelsMask->total() * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&d_filterKernel, m_filterPatameter.m_filterKnernel.total() * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_filterKernel, m_filterPatameter.m_filterKnernel.data, 
                          m_filterPatameter.m_filterKnernel.total() * sizeof(float), hipMemcpyHostToDevice));

    FilterParameterDevice filterParamDevice = { d_validNeighborPixelsNum, d_validPixelsMask, d_filterKernel };
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_filterPatameterDevice), &filterParamDevice, sizeof(FilterParameterDevice)));
	
	    // 1. 主机端分配 RanderMapPatch 数组
		RanderMapPatch* h_ppRanderMapPatch = new RanderMapPatch[m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum];

		// 2. 为每个 RanderMapPatch 的 simg 分配设备内存
		float** h_simgDevice = new float*[m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum];
		for (int i = 0; i < m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum; ++i) {
			CUDA_CHECK(hipMalloc((void**)&h_simgDevice[i], DEST_WIDTH * DEST_HEIGHT * 3 * sizeof(float)));
			h_ppRanderMapPatch[i].simg = h_simgDevice[i]; // 将设备内存地址存储在主机端的 RanderMapPatch 中
		}
	
		// 3. 设备端分配 RanderMapPatch 数组
		CUDA_CHECK(hipMalloc((void**)&d_ppRanderMapPatch, m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * sizeof(RanderMapPatch)));
	
		// 4. 复制结构体数组到设备端
		CUDA_CHECK(hipMemcpy(d_ppRanderMapPatch, h_ppRanderMapPatch, m_rawImageParameter.m_yLensNum * m_rawImageParameter.m_xLensNum * sizeof(RanderMapPatch), hipMemcpyHostToDevice));
	
	
		// 6. 释放主机端数据
		delete[] h_ppRanderMapPatch;
		delete[] h_simgDevice;
	
}

void DataParameter::UpdateImgToGPU()
{
    // 确保输入图像和裁剪后的图像是连续的
    cv::Mat inputImgContinuous = m_inputImg.isContinuous() ? m_inputImg : m_inputImg.clone();
    cv::Mat inputImgRecContinuous = m_inputImgRec.isContinuous() ? m_inputImgRec : m_inputImgRec.clone();

    // 计算图像的字节大小
    size_t inputImgSize = inputImgContinuous.total() * inputImgContinuous.elemSize();
    size_t inputImgRecSize = inputImgRecContinuous.total() * inputImgRecContinuous.elemSize();

    // 为输入图像分配内存
	CUDA_CHECK(hipMalloc((void**)&d_inputImg, inputImgSize));

	// 为裁剪后的图像分配内存
	CUDA_CHECK(hipMalloc((void**)&d_inputImgRec, inputImgRecSize));

    // 将输入图像数据从主机复制到设备
    CUDA_CHECK(hipMemcpy(d_inputImg, inputImgContinuous.ptr<float>(0), inputImgSize, hipMemcpyHostToDevice));

    // 将裁剪后的图像数据从主机复制到设备
    CUDA_CHECK(hipMemcpy(d_inputImgRec, inputImgRecContinuous.ptr<float>(0), inputImgRecSize, hipMemcpyHostToDevice));

    // 将设备指针复制到 __device__ 变量
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_inputImg), &d_inputImg, sizeof(float*)));
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_inputImgRec), &d_inputImgRec, sizeof(float*)));
}
	


DataParameter::DataParameter()
{
	m_rawImageParameter.m_yCenterBeginOffset = 2;
	m_rawImageParameter.m_xCenterBeginOffset = 2;
	m_rawImageParameter.m_yCenterEndOffset = 2;
	m_rawImageParameter.m_xCenterEndOffset = 2;
	m_filterRadius = 4;
	m_microImageParameter.m_circleDiameter = 34.0;
	m_microImageParameter.m_circleNarrow = 1.5;
	m_disparityParameter.m_dispMin = 5;
	m_disparityParameter.m_dispMax = 13;
	m_disparityParameter.m_dispStep = 0.5;
	m_microImageParameter.m_ppLensCenterPoints = nullptr;
	m_microImageParameter.m_ppPixelsMappingSet = nullptr;
	m_microImageParameter.m_ppMatchNeighborLens = nullptr;
	m_filterPatameter.m_pValidNeighborPixelsNum = nullptr;
	m_filterPatameter.m_pValidPixelsMask = nullptr;
}

DataParameter::DataParameter(std::string dataFolderName, std::string centerPointFileName, std::string inputImgName,
	int yCenterBeginOffset, int xCenterBeginOffset, int yCenterEndOffset, int xCenterEndOffset,
	int filterRadius, float circleDiameter, float circleNarrow, int dispMin, int dispMax, float dispStep)
{
	init(dataFolderName, centerPointFileName, inputImgName, yCenterBeginOffset, xCenterBeginOffset, yCenterEndOffset, xCenterEndOffset,
		filterRadius, circleDiameter, circleNarrow, dispMin, dispMax, dispStep);
}

DataParameter::~DataParameter()//�����������������ڽ������Զ�����
{
	if (m_microImageParameter.m_ppLensCenterPoints)
	{
		for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
			delete[]m_microImageParameter.m_ppLensCenterPoints[y];
		delete[]m_microImageParameter.m_ppLensCenterPoints;
	}
	if (m_microImageParameter.m_ppPixelsMappingSet)
	{
		for (int y = 0; y < m_rawImageParameter.m_srcImgHeight; y++)
			delete[]m_microImageParameter.m_ppPixelsMappingSet[y];
		delete[]m_microImageParameter.m_ppPixelsMappingSet;
	}
	if (m_microImageParameter.m_ppMatchNeighborLens)
	{
		for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
			delete[]m_microImageParameter.m_ppMatchNeighborLens[y];
		delete[]m_microImageParameter.m_ppMatchNeighborLens;
	}
	if (m_filterPatameter.m_pValidPixelsMask)
	{
		delete m_filterPatameter.m_pValidPixelsMask;
	}
	if (m_filterPatameter.m_pValidNeighborPixelsNum)
	{
		delete m_filterPatameter.m_pValidNeighborPixelsNum;
	}
}

void DataParameter::init(std::string dataFolderName, std::string centerPointFileName, std::string inputImgName,
	int yCenterBeginOffset, int xCenterBeginOffset, int yCenterEndOffset, int xCenterEndOffset,
	int filterRadius, float circleDiameter, float circleNarrow, int dispMin, int dispMax, float dispStep)
{
	lensCenterPointsInit(dataFolderName, centerPointFileName);
	validLensCenterInit(yCenterBeginOffset, xCenterBeginOffset, yCenterEndOffset, xCenterEndOffset);
	imageBaseMessageInit(inputImgName, filterRadius, circleDiameter, circleNarrow, dispMin, dispMax, dispStep);
	generatePixelsMappingSet();
	generateNeighborCenterPoints();
	generateValidPoints();
	mapToGPU();

	std::cout << "DataParameter::init final!" << std::endl;
}

void DataParameter::lensCenterPointsInit(std::string dataFolderPath, std::string centerPointFileName)
{//͸�����ĵ��ʼ��
	m_folderPath = dataFolderPath;
	std::string centerFileTxtName = dataFolderPath + "/" + centerPointFileName;
	std::ifstream ifs;
	ifs.open(centerFileTxtName, std::ifstream::in);
	ifs >> m_rawImageParameter.m_yLensNum >> m_rawImageParameter.m_xLensNum;//���ļ��ж�ȡ͸�������� m_yLensNum ������ m_xLensNum�����洢�� m_rawImageParameter �ṹ���С�
	m_microImageParameter.m_ppLensCenterPoints = new Point2d *[m_rawImageParameter.m_yLensNum];
	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
		m_microImageParameter.m_ppLensCenterPoints[y] = new Point2d[m_rawImageParameter.m_xLensNum];

	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
		for (int x = 0; x < m_rawImageParameter.m_xLensNum; x++)
			ifs >> m_microImageParameter.m_ppLensCenterPoints[y][x].y >> m_microImageParameter.m_ppLensCenterPoints[y][x].x;

	ifs.close();
	float x1 = m_microImageParameter.m_ppLensCenterPoints[0][0].x; //��0����ߵ�һ��x����
	float x2 = m_microImageParameter.m_ppLensCenterPoints[1][0].x; //��1����ߵ�һ��x����

	if (x1 > x2) m_lensArrageMode = 0; //��0�е�һ��Բ������ 
	else m_lensArrageMode = 1; //��0�е�һ��Բ����͹��
	std::cout << "lensCenterPointsInit final!" << std::endl;
}

void DataParameter::validLensCenterInit(int yCenterBeginOffset, int xCenterBeginOffset, int yCenterEndOffset, int xCenterEndOffset)
{//�����Щ͸��������Ч
	m_rawImageParameter.m_yCenterBeginOffset = yCenterBeginOffset;
	m_rawImageParameter.m_xCenterBeginOffset = xCenterBeginOffset;
	m_rawImageParameter.m_yCenterEndOffset = yCenterEndOffset;
	m_rawImageParameter.m_xCenterEndOffset = xCenterEndOffset;

	Point2d &topLeftCenterPos = m_microImageParameter.m_ppLensCenterPoints[yCenterBeginOffset - 1][xCenterBeginOffset - 1];
	Point2d &topRightCenterPos = m_microImageParameter.m_ppLensCenterPoints[yCenterBeginOffset - 1][m_rawImageParameter.m_xLensNum - xCenterEndOffset];
	Point2d &belowLeftCenterPos = m_microImageParameter.m_ppLensCenterPoints[m_rawImageParameter.m_yLensNum - yCenterEndOffset][xCenterBeginOffset - 1];
	Point2d &belowRightCenterPos = m_microImageParameter.m_ppLensCenterPoints[m_rawImageParameter.m_yLensNum - yCenterEndOffset][m_rawImageParameter.m_xLensNum - xCenterEndOffset];

	double left = std::min(topLeftCenterPos.x, belowLeftCenterPos.x);
	double right = std::max(topRightCenterPos.x, belowRightCenterPos.x);
	double top = std::min(topLeftCenterPos.y, topRightCenterPos.y);
	double below = std::max(belowLeftCenterPos.y, belowRightCenterPos.y);

	m_rawImageParameter.m_yPixelBeginOffset = top;
	m_rawImageParameter.m_xPixelBeginOffset = left;

	m_rawImageParameter.m_recImgHeight = below - top + 1;
	m_rawImageParameter.m_recImgWidth = right - left + 1;
	cout<<"validLensCenterInit_final"<<endl;
}

void DataParameter::imageBaseMessageInit(std::string inputImgName, int filterRadius, float circleDiameter,
	float circleNarrow, int dispMin, int dispMax, float dispStep)
{//ͼ�������Ϣ���Ӳ���Ϣ��ʼ��
	cout<<"imageBaseMessageInit_start"<<endl;
	m_filterRadius = filterRadius;//�˲��뾶
	m_microImageParameter.m_circleDiameter = circleDiameter;//԰ֱ��
	m_microImageParameter.m_circleNarrow = circleNarrow;//Բ�뾶��Сֵ
	m_disparityParameter.m_dispMin = dispMin;
	m_disparityParameter.m_dispMax = dispMax;
	m_disparityParameter.m_dispStep = dispStep;
	m_disparityParameter.m_disNum = double(dispMax - dispMin) / m_disparityParameter.m_dispStep; //�Ӳ�label��Ŀ
	m_filterPatameter.m_filterKnernel = cv::Mat::ones(2 * filterRadius + 1, 2 * filterRadius + 1, CV_32FC1);//�����˲��뾶���������
	m_microImageParameter.m_radiusDisEqu = (circleDiameter / 2 - m_microImageParameter.m_circleNarrow)*(circleDiameter / 2 - m_microImageParameter.m_circleNarrow);
	std::string inputImagePath = m_folderPath + "/" + inputImgName;
	cout<<"m_folderPath："<<m_folderPath<<endl;
	cout<<"inputImagePath："<<inputImagePath<<endl;
	
	m_inputImg = imread(inputImagePath, IMREAD_COLOR);
	
	//m_inputImg = imread(inputImagePath, IMREAD_GRAYSCALE);
	m_rawImageParameter.m_srcImgWidth = m_inputImg.cols;
	m_rawImageParameter.m_srcImgHeight = m_inputImg.rows;


	m_inputImgRec = m_inputImg(cv::Rect(m_rawImageParameter.m_xPixelBeginOffset, m_rawImageParameter.m_yPixelBeginOffset, 
		m_rawImageParameter.m_recImgWidth, m_rawImageParameter.m_recImgHeight)).clone();

	UpdateImgToGPU();

	std::string recImageStore = m_folderPath + "/" + "srcImgRec.png";
	imwrite(recImageStore, m_inputImgRec);
	cout<<"imageBaseMessageInit_final"<<endl;
}

void DataParameter::generatePixelsMappingSet()
{//�������ص�ӳ�伯��
	m_microImageParameter.m_ppPixelsMappingSet = new int *[m_rawImageParameter.m_srcImgHeight];
	for (int y = 0; y < m_rawImageParameter.m_srcImgHeight; y++)
	{
		m_microImageParameter.m_ppPixelsMappingSet[y] = new int[m_rawImageParameter.m_srcImgWidth];
		memset(m_microImageParameter.m_ppPixelsMappingSet[y], -1, m_rawImageParameter.m_srcImgWidth*sizeof(int));
	}

	//�����Щ��������Ч��
	m_filterPatameter.m_pValidPixelsMask = new cv::Mat;
	*m_filterPatameter.m_pValidPixelsMask = Mat::zeros(m_rawImageParameter.m_srcImgHeight, m_rawImageParameter.m_srcImgWidth, CV_32FC1);
//#pragma omp parallel for 
	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
		for (int x = 0; x < m_rawImageParameter.m_xLensNum; x++)
			generatePixelsMappingSet(y, x);

	std::cout << "generatePixelsMappingSet final!" << std::endl;
}

void DataParameter::generatePixelsMappingSet(int y, int x)
{//�������ص�ӳ�伯��--����ÿ����ͼ���cost
	Point2d &centerPos = m_microImageParameter.m_ppLensCenterPoints[y][x];
	//������ǰ͸������λ�ø���������
	for (int py = centerPos.y - m_microImageParameter.m_circleDiameter / 2 + m_microImageParameter.m_circleNarrow; 
		py <= centerPos.y + m_microImageParameter.m_circleDiameter / 2 - m_microImageParameter.m_circleNarrow; py++)
	{
		//�����������Ϊ1������ô�����������Ч
		float *yDataRowsMask = (float *)(*m_filterPatameter.m_pValidPixelsMask).ptr<float *>(py);
		for (int px = centerPos.x - m_microImageParameter.m_circleDiameter / 2 + m_microImageParameter.m_circleNarrow; 
			px <= centerPos.x + m_microImageParameter.m_circleDiameter / 2 - m_microImageParameter.m_circleNarrow; px++)
		{
			//΢͸��������
			int lens_num = y*m_rawImageParameter.m_xLensNum + x;
			//��������Ƿ���Բ��
			if ((centerPos.y - py)*(centerPos.y - py) + (centerPos.x - px)*(centerPos.x - px) <= m_microImageParameter.m_radiusDisEqu)
			{
				m_microImageParameter.m_ppPixelsMappingSet[py][px] = lens_num;
				yDataRowsMask[px] = 1.0;
			}
		}
	}
}

void DataParameter::generateNeighborCenterPoints()
{//����ÿ�����ĵ���Χ�Ķ�Ӧ���ĵ�
	//ÿ��͸�����ĵ���ھ����ĵ���Ϣ
	m_microImageParameter.m_ppMatchNeighborLens = new MatchNeighborLens **[m_rawImageParameter.m_yLensNum];
	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
	{
		m_microImageParameter.m_ppMatchNeighborLens[y] = new MatchNeighborLens *[m_rawImageParameter.m_xLensNum];
		for (int x = 0; x < m_rawImageParameter.m_xLensNum; x++)
			m_microImageParameter.m_ppMatchNeighborLens[y][x] = new MatchNeighborLens[NEIGHBOR_MATCH_LENS_NUM];
	}

//#pragma omp parallel for 
	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
		for (int x = 0; x < m_rawImageParameter.m_xLensNum; x++)
			generateNeighborCenterPoints(y, x);

	std::cout << "generateNeighborCenterPoints final!" << std::endl;
}

void DataParameter::generateNeighborCenterPoints(int y, int x)
{//����ÿ�����ĵ���Χ�Ķ�Ӧ���ĵ�
	int parityFlag = y & 1;//������Ϊ1��ż����Ϊ0
	int x_shift = 0;
	if (m_lensArrageMode ^ parityFlag){
		//��͹��ż���У������󰼣�������
		x_shift = -1;
	}
	else{
		//��͹�������У������󰼣�ż����
		x_shift = 1;
	}

	int numCount = 0;
	Point2d &curCenterPoint = m_microImageParameter.m_ppLensCenterPoints[y][x];
	double cy, cx, dis;

	
	if (y - 1 >= 0){//�Ϸ��ھӣ��൱���������������������������
		cy = m_microImageParameter.m_ppLensCenterPoints[y - 1][x].y;
		cx = m_microImageParameter.m_ppLensCenterPoints[y - 1][x].x;
		dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
		m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, (y - 1)*m_rawImageParameter.m_xLensNum + x);

		if ((x_shift < 0 && x - 1 >= 0) || (x_shift > 0 && x + 1 < m_rawImageParameter.m_xLensNum)){
			cy = m_microImageParameter.m_ppLensCenterPoints[y - 1][x + x_shift].y;
			cx = m_microImageParameter.m_ppLensCenterPoints[y - 1][x + x_shift].x;
			dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
			m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, (y - 1)*m_rawImageParameter.m_xLensNum + x + x_shift);
		}
	}

	if (y + 1 < m_rawImageParameter.m_yLensNum){//�·��ھ�
		cy = m_microImageParameter.m_ppLensCenterPoints[y + 1][x].y;
		cx = m_microImageParameter.m_ppLensCenterPoints[y + 1][x].x;
		dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
		m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, (y + 1)*m_rawImageParameter.m_xLensNum + x);

		if ((x_shift < 0 && x - 1 >= 0) || (x_shift > 0 && x + 1 < m_rawImageParameter.m_xLensNum)){
			cy = m_microImageParameter.m_ppLensCenterPoints[y + 1][x + x_shift].y;
			cx = m_microImageParameter.m_ppLensCenterPoints[y + 1][x + x_shift].x;
			dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
			m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, (y + 1)*m_rawImageParameter.m_xLensNum + x + x_shift);
		}
	}

	if (x - 1 >= 0){//����ھ�
		cy = m_microImageParameter.m_ppLensCenterPoints[y][x - 1].y;
		cx = m_microImageParameter.m_ppLensCenterPoints[y][x - 1].x;
		dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
		m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, y*m_rawImageParameter.m_xLensNum + x - 1);
	}

	if (x + 1 < m_rawImageParameter.m_xLensNum){//�Ҳ��ھ�
		cy = m_microImageParameter.m_ppLensCenterPoints[y][x + 1].y;
		cx = m_microImageParameter.m_ppLensCenterPoints[y][x + 1].x;
		dis = sqrt((cy - curCenterPoint.y)*(cy - curCenterPoint.y) + (cx - curCenterPoint.x)*(cx - curCenterPoint.x));
		m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(cy, cx, dis, y*m_rawImageParameter.m_xLensNum + x + 1);
	}

	if (numCount < NEIGHBOR_MATCH_LENS_NUM)//�����������Ϣ
		m_microImageParameter.m_ppMatchNeighborLens[y][x][numCount++] = MatchNeighborLens(-1, -1, -1, -1);
	
}

void DataParameter::generateValidPoints()
{//����ÿ��СԲ�����أ�����Χ��Ч�����
	m_filterPatameter.m_pValidNeighborPixelsNum = new cv::Mat;
	*m_filterPatameter.m_pValidNeighborPixelsNum = cv::Mat::zeros(m_rawImageParameter.m_srcImgHeight, m_rawImageParameter.m_srcImgWidth, CV_32FC1);

//#pragma omp parallel for
	for (int y = 0; y < m_rawImageParameter.m_yLensNum; y++)
		for (int x = 0; x < m_rawImageParameter.m_xLensNum; x++)
			generateValidPoints(y, x);

	validPointsBoundaryRepair();
	std::cout << "generateValidPoints final!" << std::endl;
}

void DataParameter::generateValidPoints(int y, int x)
{//����ÿ��СԲ�����أ�����Χ��Ч�����
	Point2d &curCenterPos = m_microImageParameter.m_ppLensCenterPoints[y][x];
	int x_begin = curCenterPos.x - m_microImageParameter.m_circleDiameter / 2 + m_microImageParameter.m_circleNarrow;
	int y_begin = curCenterPos.y - m_microImageParameter.m_circleDiameter / 2 + m_microImageParameter.m_circleNarrow;
	int x_end = curCenterPos.x + m_microImageParameter.m_circleDiameter / 2 - m_microImageParameter.m_circleNarrow;
	int y_end = curCenterPos.y + m_microImageParameter.m_circleDiameter / 2 - m_microImageParameter.m_circleNarrow;

	//��ȡСԲ����
	cv::Mat srcCost = (*m_filterPatameter.m_pValidPixelsMask)(cv::Rect(x_begin, y_begin, x_end - x_begin + 1, y_end - y_begin + 1));
	cv::Mat destCost = (*m_filterPatameter.m_pValidNeighborPixelsNum)(cv::Rect(x_begin, y_begin, x_end - x_begin + 1, y_end - y_begin + 1));



	cv::filter2D(srcCost, destCost, -1, m_filterPatameter.m_filterKnernel, cv::Point(-1, -1), 0, BORDER_CONSTANT);
	destCost = destCost.mul(srcCost);//������ЧԪ�أ�ͳ����Ч�ھ���
}

void DataParameter::validPointsBoundaryRepair()
{//����Ч��ı߽��������ؽ�����1��������ֹ���ֳ�0����
//#pragma omp parallel for
	for (int py = 0; py < m_rawImageParameter.m_srcImgHeight; py++)
	{
		float *yDataRows = (float *)(*m_filterPatameter.m_pValidNeighborPixelsNum).ptr<float *>(py);
		for (int px = 0; px < m_rawImageParameter.m_srcImgWidth; px++)
		{
			if (yDataRows[px] < 0.1) // �������Ч�����ٻ� >=1
				yDataRows[px] = 1.0;
		}
	}
}