#include "hip/hip_runtime.h"
#include "ImageRander.h"
#include "DataParameter.cuh"
#include <iomanip>

using namespace std;
using namespace cv;



ImageRander::ImageRander()
{

}

ImageRander::~ImageRander()
{

}


int isCalWH = 0;
 float *d_randerMap, *d_randerCount;
 int h_randerMapWidth , h_randerMapHeight;
 int randerMapWidthVal, randerMapHeightVal;
 int randerMapWidthVal_;
 int x_left, x_right, y_top, y_below;
__global__ void computeLensMeanDispKernel(MicroImageParameterDevice* d_microImageParameter,float* d_rawDisp)
{
    // 获取当前线程的坐标
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在有效范围内
    if (y >= d_rawImageParameter.m_yCenterBeginOffset && y < d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset &&
        x >= d_rawImageParameter.m_xCenterBeginOffset && x < d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset)
    {
        // 计算当前中心点坐标
        CudaPoint2f curCenterPos = CudaPoint2f(d_microImageParameter->m_ppLensCenterPoints[y * d_rawImageParameter.m_xLensNum + x].x, d_microImageParameter->m_ppLensCenterPoints[y * d_rawImageParameter.m_xLensNum + x].y);
       // printf("x:%d y:%d sx:%f sy:%f\n",x,y,curCenterPos.x,curCenterPos.y);
        int x_begin = curCenterPos.x - d_rawImageParameter.m_xPixelBeginOffset - d_meanDispLenRadius;
        int y_begin = curCenterPos.y - d_rawImageParameter.m_yPixelBeginOffset - d_meanDispLenRadius;
        // 计算区域的宽度和高度
        int rectWidth = d_meanDispLenRadius * 2 + 1;
        int rectHeight = d_meanDispLenRadius * 2 + 1;

        // 计算该区域的均值
        float sum = 0.0f;
        int count = 0;

        for (int dy = 0; dy < rectHeight; dy++) {
            for (int dx = 0; dx < rectWidth; dx++) {
                int globalX = x_begin + dx;
                int globalY = y_begin + dy;

                // 确保访问的坐标在有效范围内
                if (globalX >= 0 && globalX < d_rawImageParameter.m_recImgWidth &&
                    globalY >= 0 && globalY < d_rawImageParameter.m_recImgHeight)
                {
                    sum += d_rawDisp[globalY * d_rawImageParameter.m_recImgWidth + globalX] *255 * d_disparityParameter.m_dispStep + d_disparityParameter.m_dispMin;
                    count++;
                }
            }
        }

        float meanDisp = sum / count ;
        d_ppLensMeanDisp[y * d_rawImageParameter.m_xLensNum + x] = fmax(meanDisp, 9.0f);

    }
}





void ImageRander::imageRanderWithOutMask(const DataParameter &dataParameter)
{
    RawImageParameter rawImageParameter = dataParameter.getRawImageParameter();
    MicroImageParameter microImageParameter = dataParameter.getMicroImageParameter();
    DisparityParameter disparityParameter = dataParameter.getDisparityParameter();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Process patch kernel
    hipEventRecord(start);

        // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((rawImageParameter.m_recImgWidth + blockSize.x - 1) / blockSize.x, 
                      (rawImageParameter.m_recImgHeight + blockSize.y - 1) / blockSize.y);

        blockSize = dim3(32, 32);
        gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, 
                      (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);

    computeLensMeanDispKernel<<<gridSize, blockSize>>>(d_microImageParameter,d_rawDisp);

    // Check for any errors during kernel launch
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Compute Lens Mean Disp Kernel Time: %f ms\n", ms);

    



    hipMemset(d_randerMap, 0, randerMapWidthVal_ * randerMapHeightVal * 3 * sizeof(float));
    imageRander(rawImageParameter, microImageParameter,d_inputImgRec,3);
    saveThreeChannelGpuMemoryAsImage(d_randerMap,  randerMapWidthVal_,randerMapHeightVal, "./res/randerSceneMap.bmp");
    hipMemset(d_randerMap, 0, randerMapWidthVal_ * randerMapHeightVal * 3 * sizeof(float));
    imageRander(rawImageParameter, microImageParameter,d_rawDisp,1);
    saveSingleChannelGpuMemoryAsImage(d_randerMap, randerMapWidthVal_,randerMapHeightVal, "./res/randerDisMap.bmp");
    //保存图像会耗费约40ms时间
}


__global__ void accumulateKernel(
    RanderMapPatch* d_ppRanderMapPatch,
    float* d_randerMap,
    float* d_randerCount,
    int patchWidth,
    int patchHeight,
    int channels,
    int* sy_begin,
    int* sx_begin,
    int randerMapWidth,
    int randerMapHeight)
{
    // 当前 patch 坐标
    int patchX = blockIdx.x;
    int patchY = blockIdx.y;

    if (patchX >= d_rawImageParameter.m_xLensNum || patchY >= d_rawImageParameter.m_yLensNum)
        return;

    RanderMapPatch patch = d_ppRanderMapPatch[patchY * d_rawImageParameter.m_xLensNum + patchX];

    int sy_b = patch.sy - patchHeight / 2 - sy_begin[0];
    int sx_b = patch.sx - patchWidth  / 2 - sx_begin[0];

    // 当前线程的处理起始坐标
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int c = threadIdx.z;

    int stride_x = blockDim.x;
    int stride_y = blockDim.y;


    if (c >= channels)
        return;

    for (int py = ty; py < patchHeight; py += stride_y)
    {
        for (int px = tx; px < patchWidth; px += stride_x)
        {

            int rander_x = sx_b + px;
            int rander_y = sy_b + py;

            if (rander_x >= 0 && rander_x < randerMapWidth &&
                rander_y >= 0 && rander_y < randerMapHeight)
            {
                int simgIdx = (py * patchWidth + px) * channels + c;
                int dstIdx  = (rander_y * randerMapWidth + rander_x) * channels + c;

                float val = patch.simg[simgIdx];

                atomicAdd(&d_randerMap[dstIdx], val);

                if (c == 0)
                    atomicAdd(&d_randerCount[rander_y * randerMapWidth + rander_x], 1.0f);
            }
        }
    }
}

__global__ void normalizeKernel(float* d_randerMap,float* d_randerCount,int channels,int d_randerMapWidth,int d_randerMapHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < d_randerMapWidth && y < d_randerMapHeight)
    {
        for (int c = 0; c < channels; ++c)
        {
            int idx = (y * d_randerMapWidth + x) * channels + c;
            if (d_randerCount[y * d_randerMapWidth + x] > 0)
            {
                d_randerMap[idx] /= d_randerCount[y * d_randerMapWidth + x];
            }
        }
        
    }
}


__global__ void computeBoundaryKernel(RanderMapPatch* d_ppRanderMapPatch,
    int DEST_WIDTH_, int DEST_HEIGHT_,int* d_sx_begin, int* d_sy_begin, int* d_sx_end, int* d_sy_end)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset && y < d_rawImageParameter.m_yLensNum  - d_rawImageParameter.m_yCenterEndOffset &&
        x >= d_rawImageParameter.m_xCenterBeginOffset && y >= d_rawImageParameter.m_yCenterBeginOffset) // Ensure x and y are within valid bounds
    {
        // 使用一维数组访问
        int sy = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sy;
        int sx = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sx;
       // printf("x:%d y:%d sx:%d sy:%d\n",x,y,sx,sy);
        atomicMin(d_sx_begin, sx - DEST_WIDTH_ / 2);
        atomicMin(d_sy_begin, sy - DEST_HEIGHT_ / 2);
        atomicMax(d_sx_end, sx + DEST_WIDTH_ / 2);
        atomicMax(d_sy_end, sy + DEST_HEIGHT_ / 2);
    }
}


__global__ void processPatchKernel(MicroImageParameterDevice* d_microImageParameter, 
    RanderMapPatch* d_ppRanderMapPatch, 
    float* d_input,
    int patchWidth, int patchHeight, int Channels)
{
    // 当前处理的 patch 位置（一个线程块处理一个 patch）
    int patchX = blockIdx.x;
    int patchY = blockIdx.y;

    int c = threadIdx.z;

    // 每个线程从哪个像素开始
    int i_start = threadIdx.x;
    int j_start = threadIdx.y;

    // 每个线程跨步处理多个像素
    int stride_i = blockDim.x;
    int stride_j = blockDim.y;

    if (c >= Channels) return;

    // 获取 patch 相关信息
    int xAdjusted = patchX + d_rawImageParameter.m_xCenterBeginOffset;
    int yAdjusted = patchY + d_rawImageParameter.m_yCenterBeginOffset;

    if (xAdjusted >= d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset ||
    yAdjusted >= d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset)
    return;

    CudaPoint2f curCenterPos = d_microImageParameter->m_ppLensCenterPoints[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted];
    int blockSize = fabsf(roundf(d_ppLensMeanDisp[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted]));

    int starty = max(static_cast<int>(curCenterPos.y - blockSize / 2 - d_rawImageParameter.m_yPixelBeginOffset), 0);
    int startx = max(static_cast<int>(curCenterPos.x - blockSize / 2 - d_rawImageParameter.m_xPixelBeginOffset), 0);

    d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].sy = int(curCenterPos.y);
    d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].sx = int(curCenterPos.x);

    float* d_srcImg = d_input + (starty * d_rawImageParameter.m_recImgWidth + startx) * Channels;
    float* d_simg = d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].simg;




    int imageStride = d_rawImageParameter.m_recImgWidth;  // 原图宽度

    for (int j = j_start; j < patchHeight; j += stride_j) {
        for (int i = i_start; i < patchWidth; i += stride_i) {
            float fx = (float)i / (patchWidth - 1) * (blockSize - 1);
            float fy = (float)j / (patchHeight - 1) * (blockSize - 1);
            int ix = (int)fx;
            int iy = (int)fy;
            float wx = fx - ix;
            float wy = fy - iy;
    
            // 全局坐标（以原图为基准）
            int global_x = startx + ix;
            int global_y = starty + iy;
    
            // 插值使用原图内存访问
            float top_left     = d_input[(global_y * imageStride + global_x) * Channels + c];
            float top_right    = d_input[(global_y * imageStride + global_x + 1) * Channels + c];
            float bottom_left  = d_input[((global_y + 1) * imageStride + global_x) * Channels + c];
            float bottom_right = d_input[((global_y + 1) * imageStride + global_x + 1) * Channels + c];
    
            float interpolated = (1 - wx) * (1 - wy) * top_left +
                                 wx * (1 - wy) * top_right +
                                 (1 - wx) * wy * bottom_left +
                                 wx * wy * bottom_right;
    
            // 写入输出 patch（局部 patch 图像）
            //d_simg[(j * patchWidth + i) * Channels + c] = interpolated;
            int flip_x = patchWidth - i - 1;  // 水平翻转
            int flip_y = patchHeight - j - 1; // 垂直翻转

            d_simg[(flip_y * patchWidth + flip_x) * Channels + c] = interpolated;
        }
    }
    

}


__global__ void repairImageKernel(
    const RawImageParameter d_rawImageParameter,
    RanderMapPatch* patches,
    int* sx_begin, int* sy_begin,
    int* x_left, int* x_right,
    int* y_top, int* y_below,
    int patch_width, int patch_height)
{
// 计算当前线程的x、y坐标
int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

    // 计算有效的边界范围
    int left   = d_rawImageParameter.m_xCenterBeginOffset;
    int right  = d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset - 1;
    int top    = d_rawImageParameter.m_yCenterBeginOffset;
    int below  = d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset - 1;

    // 判断当前线程是否在有效区域内，如果在，则进行计算
    if (x >= left && x <= right && y >= top && y <= below)
    {
    int index = y * d_rawImageParameter.m_xLensNum + x;
    int sx = patches[index].sx;
    int sy = patches[index].sy;

    // 计算patch的边界
    int patch_x_left   = sx - patch_width / 2 - sx_begin[0];
    int patch_x_right  = sx + patch_width / 2 - sx_begin[0];
    int patch_y_top    = sy - patch_height / 2 - sy_begin[0];
    int patch_y_below  = sy + patch_height / 2 - sy_begin[0];

    // 使用原子操作来更新全局最小/最大值
    atomicMin(x_left, patch_x_left);
    atomicMax(x_right, patch_x_right);
    atomicMin(y_top, patch_y_top);
    atomicMax(y_below, patch_y_below);
    }
}








void ImageRander::imageRander(const RawImageParameter &rawImageParameter, 
    const MicroImageParameter &microImageParameter,float* d_input,int Channels)
{
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Process patch kernel
    hipEventRecord(start); 
    dim3 blockSize(16, 16, Channels);  

    // 图像中 patch 的总数量（X × Y）
    dim3 gridSize(
        rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterBeginOffset - rawImageParameter.m_xCenterEndOffset,
        rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterBeginOffset - rawImageParameter.m_yCenterEndOffset
    );
    
    // 启动 kernel
    processPatchKernel<<<gridSize, blockSize>>>(
        d_microImageParameter,
        d_ppRanderMapPatch,
        d_input,
        DEST_WIDTH,
        DEST_HEIGHT,
        Channels
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Process Patch Kernel Time: %f ms\n", ms); 

    if(!isCalWH) /*只计算一次就够*/
    {
        int h_sx_begin = INT_MAX, h_sy_begin = INT_MAX, h_sx_end = INT_MIN, h_sy_end = INT_MIN;
        CUDA_CHECK(hipMemcpy(d_sx_begin, &h_sx_begin, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sy_begin, &h_sy_begin, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sx_end, &h_sx_end, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sy_end, &h_sy_end, sizeof(int), hipMemcpyHostToDevice));
    
        // Step 2: Compute boundary kernel
        hipEventRecord(start); 
        blockSize = dim3(32, 32);
        gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
        computeBoundaryKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,DEST_WIDTH, DEST_HEIGHT,d_sx_begin,d_sy_begin,d_sx_end,d_sy_end);
        CUDA_CHECK(hipGetLastError()); 
        CUDA_CHECK(hipDeviceSynchronize());
        hipEventRecord(stop); 
        hipEventSynchronize(stop); 
        hipEventElapsedTime(&ms, start, stop);
        printf("Compute Boundary Kernel Time: %f ms\n", ms); 
    
    
        // Step 3: Compute width and height kernel
    
        CUDA_CHECK(hipMemcpy(&h_sx_begin, d_sx_begin, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sy_begin, d_sy_begin, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sx_end, d_sx_end, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sy_end, d_sy_end, sizeof(int), hipMemcpyDeviceToHost));
    
        printf("sy_begin:%d sy_end:%d sx_begin:%d sx_end:%d\n",h_sy_begin,h_sy_end,h_sx_begin,h_sx_end);
        h_randerMapWidth = h_sx_end - h_sx_begin + 1;
        h_randerMapHeight = h_sy_end - h_sy_begin + 1;

        randerMapHeightVal = h_randerMapHeight;
        randerMapWidthVal = h_randerMapWidth;
        randerMapWidthVal_ = h_randerMapWidth; 
        printf("randerMapWidth:%d randerMapHeight:%d\n",h_randerMapWidth,h_randerMapHeight);
        //1011 1866
    
        size_t randerMapSize = randerMapWidthVal_ * randerMapHeightVal * Channels * sizeof(float); 
        size_t randerCountSize = randerMapWidthVal_  * randerMapHeightVal * sizeof(float);
    
    
        CUDA_CHECK(hipMalloc(&d_randerMap, randerMapSize));
        CUDA_CHECK(hipMalloc(&d_randerCount, randerCountSize));
    
        // 初始化设备内存
        CUDA_CHECK(hipMemset(d_randerMap, 0, randerMapSize));
        CUDA_CHECK(hipMemset(d_randerCount, 0, randerCountSize));
        
    }




    // Step 5: Accumulate kernel
    hipEventRecord(start); // 记录开始时间
    blockSize = dim3(16, 16, Channels); 
    gridSize=  dim3(rawImageParameter.m_xLensNum, rawImageParameter.m_yLensNum); 

    accumulateKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,d_randerMap,d_randerCount,DEST_WIDTH, DEST_HEIGHT, Channels,d_sy_begin,d_sx_begin,randerMapWidthVal_,randerMapHeightVal); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Accumulate Kernel Time: %f ms\n", ms); 


    // Step 6: Normalize kernel
    hipEventRecord(start); 
    blockSize = dim3(32, 32);
    gridSize.x = ( randerMapWidthVal_ + blockSize.x - 1) / blockSize.x;
    gridSize.y = (randerMapHeightVal + blockSize.y - 1) / blockSize.y;
    normalizeKernel<<<gridSize, blockSize>>>(d_randerMap,d_randerCount,Channels,randerMapWidthVal_,randerMapHeightVal); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Normalize Kernel Time: %f ms\n", ms); 

    if(!isCalWH)
    {
        int* d_x_left;
        int* d_x_right;
        int* d_y_top;
        int* d_y_below;
        hipMalloc(&d_x_left, sizeof(int));
        hipMalloc(&d_x_right, sizeof(int));
        hipMalloc(&d_y_top, sizeof(int));
        hipMalloc(&d_y_below, sizeof(int));

        // 初始化边界值
        int initValue = 0;
        hipMemcpy(d_x_left, &initValue, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_x_right, &randerMapWidthVal_, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_y_top, &initValue, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_y_below, &randerMapHeightVal, sizeof(int), hipMemcpyHostToDevice);
        blockSize = dim3(32, 32);  // 这里可以根据你的硬件调整
        gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x,
                      (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
        
        repairImageKernel<<<gridSize, blockSize>>>(
            d_rawImageParameter,
            d_ppRanderMapPatch,
            d_sx_begin, d_sy_begin,
            d_x_left, d_x_right,
            d_y_top, d_y_below,
            DEST_WIDTH, DEST_HEIGHT
        );

        CUDA_CHECK(hipGetLastError()); 
        CUDA_CHECK(hipDeviceSynchronize());

        hipMemcpy(&x_left, d_x_left, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&x_right, d_x_right, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&y_top, d_y_top, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&y_below, d_y_below, sizeof(int), hipMemcpyDeviceToHost);
        isCalWH = 1 ;
    }


   
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void saveSingleChannelGpuMemoryAsImage(float* d_data, int width, int height, const std::string& filename) {
    // 分配主机内存
    float* h_data = new float[width * height];

    // 从设备内存拷贝到主机内存
    CUDA_CHECK(hipMemcpy(h_data, d_data, width * height * sizeof(float), hipMemcpyDeviceToHost));

    // 将浮点数据转换为 uchar 数据
    cv::Mat img(height, width, CV_32FC1, h_data);
	double minVal; double maxVal;
	minMaxLoc(img, &minVal, &maxVal);
    cv::Mat img_8u;
    img(cv::Rect(x_left, y_top, x_right - x_left, y_below - y_top)).convertTo(img_8u, CV_8UC1,255.0 / (maxVal - minVal), -minVal*255.0 / (maxVal - minVal)); // 将浮点值 [0, 1] 转换为 [0, 255]

    // 保存图像
    cv::imwrite(filename, img_8u);

    // 释放主机内存
    delete[] h_data;
}

void saveThreeChannelGpuMemoryAsImage(float* d_data, int width, int height, const std::string& filename) {
    // 分配主机内存
    float* h_data = new float[width * height * 3];

    // 从设备内存拷贝到主机内存
    CUDA_CHECK(hipMemcpy(h_data, d_data, width * height * 3 * sizeof(float), hipMemcpyDeviceToHost));

    // 将浮点数据转换为 uchar 数据
    cv::Mat img(height, width, CV_32FC3, h_data);
    cv::Mat img_8u;
    img(cv::Rect(x_left, y_top, x_right - x_left, y_below - y_top)).convertTo(img_8u, CV_8UC3, 255.0); // 将浮点值 [0, 1] 转换为 [0, 255]

    // 保存图像
    cv::imwrite(filename, img_8u);

    // 释放主机内存
    delete[] h_data;
}


void ImageRander::outputSparseSceneDepth(string folderName, cv::Mat &sceneSparseDepth, cv::Mat &sceneDepthMask)
{
	std::string storeName;
	ofstream ofs1;
	storeName = folderName + "/sceneInitDisp.txt";
	ofs1.open(storeName, ofstream::out);
	ofstream ofs2;
	storeName = folderName + "/sceneInitDispMask.txt";
	ofs2.open(storeName, ofstream::out);

	for (int y = 0; y < sceneSparseDepth.rows; ++y)
	{
		double *pYSceneDisp = (double *)sceneSparseDepth.ptr<double>(y);
		uchar *pYMask = (uchar *)sceneDepthMask.ptr<uchar>(y);
		for (int x = 0; x < sceneSparseDepth.cols; ++x)
		{
			ofs1 << fixed << setprecision(5) << pYSceneDisp[x] << " ";
			if (pYMask[x] == 255)
				ofs2 << 1 << " ";
			else
				ofs2 << 0 << " ";
			//ofs2 << int(pYMask[x]) << " ";
		}
		ofs1 << endl;
		ofs2 << endl;
	}
	ofs1.close();
	ofs2.close();
}