#include "hip/hip_runtime.h"
#include "ImageRander.h"
#include "DataParameter.cuh"
#include <iomanip>

using namespace std;
using namespace cv;



ImageRander::ImageRander()
{

}

ImageRander::~ImageRander()
{

}


int isCalWH = 0;
 float *d_randerMap, *d_randerCount;
 int h_randerMapWidth , h_randerMapHeight;
 int randerMapWidthVal, randerMapHeightVal;
 int randerMapWidthVal_;
__global__ void computeLensMeanDispKernel(MicroImageParameterDevice* d_microImageParameter,float* d_rawDisp)
{
    // 获取当前线程的坐标
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在有效范围内
    if (y >= d_rawImageParameter.m_yCenterBeginOffset && y < d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset &&
        x >= d_rawImageParameter.m_xCenterBeginOffset && x < d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset)
    {
        // 计算当前中心点坐标
        CudaPoint2f curCenterPos = CudaPoint2f(d_microImageParameter->m_ppLensCenterPoints[y * d_rawImageParameter.m_xLensNum + x].x, d_microImageParameter->m_ppLensCenterPoints[y * d_rawImageParameter.m_xLensNum + x].y);
       // printf("x:%d y:%d sx:%f sy:%f\n",x,y,curCenterPos.x,curCenterPos.y);
        int x_begin = curCenterPos.x - d_rawImageParameter.m_xPixelBeginOffset - d_meanDispLenRadius;
        int y_begin = curCenterPos.y - d_rawImageParameter.m_yPixelBeginOffset - d_meanDispLenRadius;
        // 计算区域的宽度和高度
        int rectWidth = d_meanDispLenRadius * 2 + 1;
        int rectHeight = d_meanDispLenRadius * 2 + 1;

        // 计算该区域的均值
        float sum = 0.0f;
        int count = 0;

        for (int dy = 0; dy < rectHeight; dy++) {
            for (int dx = 0; dx < rectWidth; dx++) {
                int globalX = x_begin + dx;
                int globalY = y_begin + dy;

                // 确保访问的坐标在有效范围内
                if (globalX >= 0 && globalX < d_rawImageParameter.m_recImgWidth &&
                    globalY >= 0 && globalY < d_rawImageParameter.m_recImgHeight)
                {
                    sum += d_rawDisp[globalY * d_rawImageParameter.m_recImgWidth + globalX] *255 * d_disparityParameter.m_dispStep + d_disparityParameter.m_dispMin;
                    count++;
                }
            }
        }

        float meanDisp = sum / count ;
        d_ppLensMeanDisp[y * d_rawImageParameter.m_xLensNum + x] = fmax(meanDisp, 9.0f);

    }
}





void ImageRander::imageRanderWithOutMask(const DataParameter &dataParameter)
{
    RawImageParameter rawImageParameter = dataParameter.getRawImageParameter();
    MicroImageParameter microImageParameter = dataParameter.getMicroImageParameter();
    DisparityParameter disparityParameter = dataParameter.getDisparityParameter();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Process patch kernel
    hipEventRecord(start);

        // Define block and grid sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((rawImageParameter.m_recImgWidth + blockSize.x - 1) / blockSize.x, 
                      (rawImageParameter.m_recImgHeight + blockSize.y - 1) / blockSize.y);

        blockSize = dim3(32, 32);
        gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, 
                      (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);

    computeLensMeanDispKernel<<<gridSize, blockSize>>>(d_microImageParameter,d_rawDisp);

    // Check for any errors during kernel launch
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Compute Lens Mean Disp Kernel Time: %f ms\n", ms);

    



    hipMemset(d_randerMap, 0, randerMapWidthVal_ * randerMapHeightVal * 3 * sizeof(float));
    imageRander(rawImageParameter, microImageParameter,d_inputImgRec,3);
    saveThreeChannelGpuMemoryAsImage(d_randerMap,  randerMapWidthVal_,randerMapHeightVal, "./res/randerSceneMap.bmp");
    hipMemset(d_randerMap, 0, randerMapWidthVal_ * randerMapHeightVal * 3 * sizeof(float));
    imageRander(rawImageParameter, microImageParameter,d_rawDisp,1);
    saveSingleChannelGpuMemoryAsImage(d_randerMap, randerMapWidthVal_,randerMapHeightVal, "./res/randerDisMap.bmp");
    //保存图像会耗费约40ms时间
}


__global__ void accumulateKernel(
    RanderMapPatch* d_ppRanderMapPatch,
    float* d_randerMap,
    float* d_randerCount,
    int patchWidth,
    int patchHeight,
    int channels,
    int* sy_begin,
    int* sx_begin,
    int randerMapWidth,
    int randerMapHeight)
{
    // 当前 patch 坐标
    int patchX = blockIdx.x;
    int patchY = blockIdx.y;

    if (patchX >= d_rawImageParameter.m_xLensNum || patchY >= d_rawImageParameter.m_yLensNum)
        return;

    RanderMapPatch patch = d_ppRanderMapPatch[patchY * d_rawImageParameter.m_xLensNum + patchX];

    int sy_b = patch.sy - patchHeight / 2 - sy_begin[0];
    int sx_b = patch.sx - patchWidth  / 2 - sx_begin[0];

    // 当前线程的处理起始坐标
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int c = threadIdx.z;

    int stride_x = blockDim.x;
    int stride_y = blockDim.y;


    if (c >= channels)
        return;

    for (int py = ty; py < patchHeight; py += stride_y)
    {
        for (int px = tx; px < patchWidth; px += stride_x)
        {

            int rander_x = sx_b + px;
            int rander_y = sy_b + py;

            if (rander_x >= 0 && rander_x < randerMapWidth &&
                rander_y >= 0 && rander_y < randerMapHeight)
            {
                int simgIdx = (py * patchWidth + px) * channels + c;
                int dstIdx  = (rander_y * randerMapWidth + rander_x) * channels + c;

                float val = patch.simg[simgIdx];

                atomicAdd(&d_randerMap[dstIdx], val);

                if (c == 0)
                    atomicAdd(&d_randerCount[rander_y * randerMapWidth + rander_x], 1.0f);
            }
        }
    }
}

__global__ void normalizeKernel(float* d_randerMap,float* d_randerCount,int channels,int d_randerMapWidth,int d_randerMapHeight)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < d_randerMapWidth && y < d_randerMapHeight)
    {
        for (int c = 0; c < channels; ++c)
        {
            int idx = (y * d_randerMapWidth + x) * channels + c;
            if (d_randerCount[y * d_randerMapWidth + x] > 0)
            {
                d_randerMap[idx] /= d_randerCount[y * d_randerMapWidth + x];
            }
        }
        
    }
}


__global__ void computeBoundaryKernel(RanderMapPatch* d_ppRanderMapPatch,
    int DEST_WIDTH_, int DEST_HEIGHT_,int* d_sx_begin, int* d_sy_begin, int* d_sx_end, int* d_sy_end)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < d_rawImageParameter.m_xLensNum && y < d_rawImageParameter.m_yLensNum &&
        x >= 0 && y >= 0) // Ensure x and y are within valid bounds
    {
        // 使用一维数组访问
        int sy = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sy;
        int sx = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sx;
       // printf("x:%d y:%d sx:%d sy:%d\n",x,y,sx,sy);
        atomicMin(d_sx_begin, sx - DEST_WIDTH_ / 2);
        atomicMin(d_sy_begin, sy - DEST_HEIGHT_ / 2);
        atomicMax(d_sx_end, sx + DEST_WIDTH_ / 2);
        atomicMax(d_sy_end, sy + DEST_HEIGHT_ / 2);
    }
}


__global__ void processPatchKernel(MicroImageParameterDevice* d_microImageParameter, 
    RanderMapPatch* d_ppRanderMapPatch, 
    float* d_input,
    int patchWidth, int patchHeight, int Channels)
{
    // 当前处理的 patch 位置（一个线程块处理一个 patch）
    int patchX = blockIdx.x;
    int patchY = blockIdx.y;

    int c = threadIdx.z;

    // 每个线程从哪个像素开始
    int i_start = threadIdx.x;
    int j_start = threadIdx.y;

    // 每个线程跨步处理多个像素
    int stride_i = blockDim.x;
    int stride_j = blockDim.y;

    if (c >= Channels) return;

    // 获取 patch 相关信息
    int xAdjusted = patchX + d_rawImageParameter.m_xCenterBeginOffset;
    int yAdjusted = patchY + d_rawImageParameter.m_yCenterBeginOffset;

    if (xAdjusted >= d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset ||
    yAdjusted >= d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset)
    return;

    CudaPoint2f curCenterPos = d_microImageParameter->m_ppLensCenterPoints[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted];
    int blockSize = fabsf(roundf(d_ppLensMeanDisp[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted]));

    int starty = max(static_cast<int>(curCenterPos.y - blockSize / 2 - d_rawImageParameter.m_yPixelBeginOffset), 0);
    int startx = max(static_cast<int>(curCenterPos.x - blockSize / 2 - d_rawImageParameter.m_xPixelBeginOffset), 0);

    d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].sy = int(curCenterPos.y);
    d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].sx = int(curCenterPos.x);

    float* d_srcImg = d_input + (starty * d_rawImageParameter.m_recImgWidth + startx) * Channels;
    float* d_simg = d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].simg;




    int imageStride = d_rawImageParameter.m_recImgWidth;  // 原图宽度

    for (int j = j_start; j < patchHeight; j += stride_j) {
        for (int i = i_start; i < patchWidth; i += stride_i) {
            float fx = (float)i / (patchWidth - 1) * (blockSize - 1);
            float fy = (float)j / (patchHeight - 1) * (blockSize - 1);
            int ix = (int)fx;
            int iy = (int)fy;
            float wx = fx - ix;
            float wy = fy - iy;
    
            // 全局坐标（以原图为基准）
            int global_x = startx + ix;
            int global_y = starty + iy;
    
            // 插值使用原图内存访问
            float top_left     = d_input[(global_y * imageStride + global_x) * Channels + c];
            float top_right    = d_input[(global_y * imageStride + global_x + 1) * Channels + c];
            float bottom_left  = d_input[((global_y + 1) * imageStride + global_x) * Channels + c];
            float bottom_right = d_input[((global_y + 1) * imageStride + global_x + 1) * Channels + c];
    
            float interpolated = (1 - wx) * (1 - wy) * top_left +
                                 wx * (1 - wy) * top_right +
                                 (1 - wx) * wy * bottom_left +
                                 wx * wy * bottom_right;
    
            // 写入输出 patch（局部 patch 图像）
            //d_simg[(j * patchWidth + i) * Channels + c] = interpolated;
            int flip_x = patchWidth - i - 1;  // 水平翻转
            int flip_y = patchHeight - j - 1; // 垂直翻转

            d_simg[(flip_y * patchWidth + flip_x) * Channels + c] = interpolated;
        }
    }
    

}


__global__ void repairImageKernel(const RawImageParameter d_rawImageParameter, 
    RanderMapPatch* d_ppRanderMapPatch, 
    int* sx_begin, int* sy_begin,
    int* d_x_left, int* d_x_right, 
    int* d_y_top, int* d_y_below,int patch_width, int patch_height)
{
    int patchY = blockIdx.y;
    int patchX = blockIdx.x;

    // 每个线程处理一个patch
    if (patchX >= d_rawImageParameter.m_xLensNum || patchY >= d_rawImageParameter.m_yLensNum) {
        return;
    }

    int left = d_rawImageParameter.m_xCenterBeginOffset;
    int right = d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset - 1;
    int top = d_rawImageParameter.m_yCenterBeginOffset;
    int below = d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset - 1;

    int patch_sx = d_ppRanderMapPatch[patchY * d_rawImageParameter.m_xLensNum + patchX].sx;
    int patch_sy = d_ppRanderMapPatch[patchY * d_rawImageParameter.m_xLensNum + patchX].sy;

    // 计算 patch 对应的区域边界
    int patch_x_left = patch_sx - patch_width / 2 - *sx_begin;
    int patch_x_right = patch_sx + patch_width / 2 - *sx_begin;
    int patch_y_top = patch_sy - patch_height / 2 - *sy_begin;
    int patch_y_below = patch_sy + patch_height / 2 - *sy_begin;

    // 使用原子操作来确保线程安全地更新最小/最大边界
    atomicMin(d_x_left, patch_x_left);
    atomicMax(d_x_right, patch_x_right);
    atomicMin(d_y_top, patch_y_top);
    atomicMax(d_y_below, patch_y_below);
}







void ImageRander::imageRander(const RawImageParameter &rawImageParameter, 
    const MicroImageParameter &microImageParameter,float* d_input,int Channels)
{
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Process patch kernel
    hipEventRecord(start); 
    dim3 blockSize(16, 16, Channels);  

    // 图像中 patch 的总数量（X × Y）
    dim3 gridSize(
        rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterBeginOffset - rawImageParameter.m_xCenterEndOffset,
        rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterBeginOffset - rawImageParameter.m_yCenterEndOffset
    );
    
    // 启动 kernel
    processPatchKernel<<<gridSize, blockSize>>>(
        d_microImageParameter,
        d_ppRanderMapPatch,
        d_input,
        DEST_WIDTH,
        DEST_HEIGHT,
        Channels
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Process Patch Kernel Time: %f ms\n", ms); 

    if(!isCalWH) /*只计算一次就够*/
    {
        int h_sx_begin = INT_MAX, h_sy_begin = INT_MAX, h_sx_end = INT_MIN, h_sy_end = INT_MIN;
        CUDA_CHECK(hipMemcpy(d_sx_begin, &h_sx_begin, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sy_begin, &h_sy_begin, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sx_end, &h_sx_end, sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_sy_end, &h_sy_end, sizeof(int), hipMemcpyHostToDevice));
    
        // Step 2: Compute boundary kernel
        hipEventRecord(start); 
        blockSize = dim3(32, 32);
        gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
        computeBoundaryKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,DEST_WIDTH, DEST_HEIGHT,d_sx_begin,d_sy_begin,d_sx_end,d_sy_end);
        CUDA_CHECK(hipGetLastError()); 
        CUDA_CHECK(hipDeviceSynchronize());
        hipEventRecord(stop); 
        hipEventSynchronize(stop); 
        hipEventElapsedTime(&ms, start, stop);
        printf("Compute Boundary Kernel Time: %f ms\n", ms); 
    
    
        // Step 3: Compute width and height kernel
    
        CUDA_CHECK(hipMemcpy(&h_sx_begin, d_sx_begin, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sy_begin, d_sy_begin, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sx_end, d_sx_end, sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(&h_sy_end, d_sy_end, sizeof(int), hipMemcpyDeviceToHost));
    
    
        h_randerMapWidth = h_sx_end - h_sx_begin + 1;
        h_randerMapHeight = h_sy_end - h_sy_begin + 1;

        randerMapHeightVal = h_randerMapHeight;
        randerMapWidthVal = h_randerMapWidth;
        randerMapWidthVal_ = h_randerMapWidth; 
        printf("randerMapWidth:%d randerMapHeight:%d\n",h_randerMapWidth,h_randerMapHeight);
        //1011 1866
    
        size_t randerMapSize = randerMapWidthVal_ * randerMapHeightVal * Channels * sizeof(float); 
        size_t randerCountSize = randerMapWidthVal_  * randerMapHeightVal * sizeof(float);
    
    
        CUDA_CHECK(hipMalloc(&d_randerMap, randerMapSize));
        CUDA_CHECK(hipMalloc(&d_randerCount, randerCountSize));
    
        // 初始化设备内存
        CUDA_CHECK(hipMemset(d_randerMap, 0, randerMapSize));
        CUDA_CHECK(hipMemset(d_randerCount, 0, randerCountSize));
        
    }




    // Step 5: Accumulate kernel
    hipEventRecord(start); // 记录开始时间
    blockSize = dim3(16, 16, Channels); 
    gridSize=  dim3(rawImageParameter.m_xLensNum, rawImageParameter.m_yLensNum); 

    accumulateKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,d_randerMap,d_randerCount,DEST_WIDTH, DEST_HEIGHT, Channels,d_sy_begin,d_sx_begin,randerMapWidthVal_,randerMapHeightVal); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Accumulate Kernel Time: %f ms\n", ms); 


    // Step 6: Normalize kernel
    hipEventRecord(start); 
    gridSize.x = ( randerMapWidthVal_ + blockSize.x - 1) / blockSize.x;
    gridSize.y = (randerMapHeightVal + blockSize.y - 1) / blockSize.y;
    normalizeKernel<<<gridSize, blockSize>>>(d_randerMap,d_randerCount,Channels,randerMapWidthVal_,randerMapHeightVal); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Normalize Kernel Time: %f ms\n", ms); 

    if(!isCalWH)
    {
        int* d_x_left;
        int* d_x_right;
        int* d_y_top;
        int* d_y_below;
        hipMalloc(&d_x_left, sizeof(int));
        hipMalloc(&d_x_right, sizeof(int));
        hipMalloc(&d_y_top, sizeof(int));
        hipMalloc(&d_y_below, sizeof(int));

        // 初始化边界值
        int initValue = 0;
        hipMemcpy(d_x_left, &initValue, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_x_right, &randerMapWidthVal_, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_y_top, &initValue, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_y_below, &randerMapHeightVal, sizeof(int), hipMemcpyHostToDevice);
        gridSize =  dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x,
        (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
        repairImageKernel<<<gridSize, blockSize>>>(d_rawImageParameter, d_ppRanderMapPatch, d_sx_begin, d_sy_begin, 
            d_x_left, d_x_right, d_y_top, d_y_below,DEST_WIDTH, DEST_HEIGHT);

        CUDA_CHECK(hipGetLastError()); 
        CUDA_CHECK(hipDeviceSynchronize());

        int x_left, x_right, y_top, y_below;
        hipMemcpy(&x_left, d_x_left, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&x_right, d_x_right, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&y_top, d_y_top, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&y_below, d_y_below, sizeof(int), hipMemcpyDeviceToHost);
        isCalWH = 1 ;
    }


   
    hipEventDestroy(start);
    hipEventDestroy(stop);
}








/*
void ImageRander::imageRander(float **ppLensMeanDisp, const RawImageParameter &rawImageParameter, const MicroImageParameter &microImageParameter, cv::Mat &randerImg, cv::Mat &destImg)
{
    RanderMapPatch **ppRanderMapPatch = new RanderMapPatch *[rawImageParameter.m_yLensNum];
    for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
        ppRanderMapPatch[i] = new RanderMapPatch[rawImageParameter.m_xLensNum];

    float *d_randerImg, *d_tmp, *d_simg;
    size_t imgSize = randerImg.rows * randerImg.cols * randerImg.channels() * sizeof(float);
    hipMalloc(&d_randerImg, imgSize);
    hipMemcpy(d_randerImg, randerImg.ptr<float>(), imgSize, hipMemcpyHostToDevice);

    size_t tmpSize = DEST_WIDTH * DEST_HEIGHT * randerImg.channels() * sizeof(float);
    hipMalloc(&d_tmp, tmpSize);
    hipMalloc(&d_simg, tmpSize);

    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            int blockSize = fabs(std::round(d_ppLensMeanDisp[y * rawImageParameter.m_xLensNum + x]));

            Point2d &curCenterPos = microImageParameter.m_ppLensCenterPoints[y][x];
            int starty = curCenterPos.y - blockSize / 2 - rawImageParameter.m_yPixelBeginOffset;
            int startx = curCenterPos.x - blockSize / 2 - rawImageParameter.m_xPixelBeginOffset;

            float *d_srcImg = d_randerImg + (starty * randerImg.cols + startx) * randerImg.channels();
            resizeCUDA(d_srcImg, d_tmp, blockSize, blockSize, DEST_WIDTH, DEST_HEIGHT, randerImg.channels());
            flipCUDA(d_tmp, d_simg, DEST_WIDTH, DEST_HEIGHT, randerImg.channels());

            ppRanderMapPatch[y][x].sy = curCenterPos.y;
            ppRanderMapPatch[y][x].sx = curCenterPos.x;
            ppRanderMapPatch[y][x].simg = d_simg;
        }
    }

    int sx_begin = INT_MAX, sy_begin = INT_MAX;
    int sx_end = INT_MIN, sy_end = INT_MIN;

    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            sy_begin = std::min(sy_begin, ppRanderMapPatch[y][x].sy - DEST_HEIGHT / 2);
            sx_begin = std::min(sx_begin, ppRanderMapPatch[y][x].sx - DEST_WIDTH / 2);
            sy_end = std::max(sy_end, ppRanderMapPatch[y][x].sy + DEST_HEIGHT / 2);
            sx_end = std::max(sx_end, ppRanderMapPatch[y][x].sx + DEST_WIDTH / 2);
        }            
    }

    int randerMapWidth = sx_end - sx_begin + 1;
    int randerMapHeight = sy_end - sy_begin + 1;

    float *d_randerMap, *d_randerCount;
    size_t randerMapSize = randerMapWidth * randerMapHeight * randerImg.channels() * sizeof(float);
    size_t randerCountSize = randerMapWidth * randerMapHeight * sizeof(float);
    hipMalloc(&d_randerMap, randerMapSize);
    hipMalloc(&d_randerCount, randerCountSize);
    hipMemset(d_randerMap, 0, randerMapSize);
    hipMemset(d_randerCount, 0, randerCountSize);

    dim3 blockSize(16, 16);
    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            int sy_b = ppRanderMapPatch[y][x].sy - DEST_HEIGHT / 2 - sy_begin;
            int sx_b = ppRanderMapPatch[y][x].sx - DEST_WIDTH / 2 - sx_begin;

            dim3 gridSize((DEST_WIDTH + blockSize.x - 1) / blockSize.x, (DEST_HEIGHT + blockSize.y - 1) / blockSize.y);
            accumulateKernel<<<gridSize, blockSize>>>(d_randerMap, d_randerCount, ppRanderMapPatch[y][x].simg, sx_b, sy_b, DEST_WIDTH, DEST_HEIGHT, randerMapWidth, randerMapHeight, randerImg.channels());
        }
    }

    dim3 gridSize((randerMapWidth + blockSize.x - 1) / blockSize.x, (randerMapHeight + blockSize.y - 1) / blockSize.y);
    normalizeKernel<<<gridSize, blockSize>>>(d_randerMap, d_randerCount, randerMapWidth, randerMapHeight, randerImg.channels());

    float *h_randerMap = new float[randerMapWidth * randerMapHeight * randerImg.channels()];
    hipMemcpy(h_randerMap, d_randerMap, randerMapSize, hipMemcpyDeviceToHost);

    destImg = cv::Mat(randerMapHeight, randerMapWidth, randerImg.type(), h_randerMap);

    hipFree(d_randerMap);
    hipFree(d_randerCount);
    hipFree(d_randerImg);
    hipFree(d_tmp);
    hipFree(d_simg);

    for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
        delete[] ppRanderMapPatch[i];
    delete[] ppRanderMapPatch;
}*/

void ImageRander::outputSparseSceneDepth(string folderName, cv::Mat &sceneSparseDepth, cv::Mat &sceneDepthMask)
{
	std::string storeName;
	ofstream ofs1;
	storeName = folderName + "/sceneInitDisp.txt";
	ofs1.open(storeName, ofstream::out);
	ofstream ofs2;
	storeName = folderName + "/sceneInitDispMask.txt";
	ofs2.open(storeName, ofstream::out);

	for (int y = 0; y < sceneSparseDepth.rows; ++y)
	{
		double *pYSceneDisp = (double *)sceneSparseDepth.ptr<double>(y);
		uchar *pYMask = (uchar *)sceneDepthMask.ptr<uchar>(y);
		for (int x = 0; x < sceneSparseDepth.cols; ++x)
		{
			ofs1 << fixed << setprecision(5) << pYSceneDisp[x] << " ";
			if (pYMask[x] == 255)
				ofs2 << 1 << " ";
			else
				ofs2 << 0 << " ";
			//ofs2 << int(pYMask[x]) << " ";
		}
		ofs1 << endl;
		ofs2 << endl;
	}
	ofs1.close();
	ofs2.close();
}