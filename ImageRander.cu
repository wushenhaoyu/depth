#include "hip/hip_runtime.h"
#include "ImageRander.h"
#include "DataParameter.cuh"
#include <iomanip>

using namespace std;
using namespace cv;



ImageRander::ImageRander()
{

}

ImageRander::~ImageRander()
{

}

/*void ImageRander::imageRanderWithMask(const DataParameter &dataParameter, cv::Mat &rawDisp, cv::Mat *confidentMask)
{//对带置信度mask的情况进行子孔径渲染
	RawImageParameter rawImageParameter = dataParameter.getRawImageParameter();
	MicroImageParameter microImageParameter = dataParameter.getMicroImageParameter();
	DisparityParameter disparityParameter = dataParameter.getDisparityParameter();

	cv::Mat tmpRawDisp = cv::Mat::zeros(rawDisp.rows, rawDisp.cols, CV_32FC1);
	rawDisp.convertTo(tmpRawDisp, CV_32F, disparityParameter.m_dispStep, disparityParameter.m_dispMin);

	float **ppLensMeanDisp = new float*[rawImageParameter.m_yLensNum];
	for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
		ppLensMeanDisp[i] = new float[rawImageParameter.m_xLensNum];

//#pragma omp parallel for
	for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
	{
		for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
		{
			Point2d &curCenterPos = microImageParameter.m_ppLensCenterPoints[y][x];
			int x_begin = curCenterPos.x - rawImageParameter.m_xPixelBeginOffset - MEAN_DISP_LEN_RADIUS;
			int y_begin = curCenterPos.y - rawImageParameter.m_yPixelBeginOffset - MEAN_DISP_LEN_RADIUS;

			cv::Mat srcCost = tmpRawDisp(cv::Rect(x_begin, y_begin, MEAN_DISP_LEN_RADIUS * 2 + 1, MEAN_DISP_LEN_RADIUS * 2 + 1));
			cv::Mat mask = (*confidentMask)(cv::Rect(x_begin, y_begin, MEAN_DISP_LEN_RADIUS * 2 + 1, MEAN_DISP_LEN_RADIUS * 2 + 1));
			ppLensMeanDisp[y][x] = std::max(cv::mean(srcCost, mask)[0], (double)(disparityParameter.m_dispMin));
		}
	}

	//渲染开始
	cv::Mat randerMapinput, randerSceneMap, finalRanderMap;
	//dataParameter.m_inputImgRec.convertTo(randerMapinput, CV_64FC3);
	dataParameter.m_inputImgRec.convertTo(randerMapinput, CV_64FC1);//转为双精度浮点数单通道
	imageRander(ppLensMeanDisp, rawImageParameter, microImageParameter, randerMapinput, randerSceneMap);
	std::string storeName = dataParameter.m_folderPath + "/randerSceneMap.bmp";
	//randerSceneMap.convertTo(finalRanderMap, CV_8UC3);
	randerSceneMap.convertTo(finalRanderMap, CV_8UC1);
	imwrite(storeName, finalRanderMap);

	//这部分用来渲染深度图像，测试速度阶段暂时注释掉
//	/*
	cv::Mat sceneDisp = cv::Mat::zeros(rawDisp.rows, rawDisp.cols, CV_64FC1);
	rawDisp.convertTo(sceneDisp, CV_64FC1, disparityParameter.m_dispStep, disparityParameter.m_dispMin);
	cv::Mat randerDispMap, randerSparseDispMap;
	imageRander(ppLensMeanDisp, rawImageParameter, microImageParameter, sceneDisp, randerDispMap);
	randerDispMap.copyTo(randerSparseDispMap);
	storeName = dataParameter.m_folderPath + "/randerDispMap.bmp";
	dispMapShowForColor(storeName, randerDispMap);

	cv::Mat rawMask, randerSceneMask;
	(*confidentMask).convertTo(rawMask, CV_64FC1);
	imageRander(ppLensMeanDisp, rawImageParameter, microImageParameter, rawMask, randerSceneMask);
	cv::Mat finalRanderMask = cv::Mat::zeros(randerSceneMask.rows, randerSceneMask.cols, CV_8UC1);
//#pragma omp parallel for
	for (int py = 0; py < randerSceneMask.rows; py++)
	{
		double *ySceneData = (double *)randerSceneMask.ptr<double>(py);
		uchar *yFinalData = (uchar *)finalRanderMask.ptr<uchar>(py);
		for (int px = 0; px < randerSceneMask.cols; px++)
		{
			if (ySceneData[px] >= 255.0)
				yFinalData[px] = 255;
		}
	}
	storeName = dataParameter.m_folderPath + "/randerSceneMask.bmp";
	imwrite(storeName, finalRanderMask);

	outputSparseSceneDepth(dataParameter.m_folderPath, randerSparseDispMap, finalRanderMask);

	double minVal, maxVal;
	minMaxLoc(randerSparseDispMap, &minVal, &maxVal);

	for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
		delete[]ppLensMeanDisp[i];
	delete[]ppLensMeanDisp;
}*/

 float *d_randerMap, *d_randerCount;
 int h_randerMapWidth , h_randerMapHeight;
__global__ void computeLensMeanDispKernel(float* d_rawDisp)
{
    // 获取当前线程的坐标
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在有效范围内
    if (y >= d_rawImageParameter.m_yCenterBeginOffset && y < d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset &&
        x >= d_rawImageParameter.m_xCenterBeginOffset && x < d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset)
    {
        // 计算当前中心点坐标
        Point2d curCenterPos = d_microImageParameter.m_ppLensCenterPoints[y * d_rawImageParameter.m_xLensNum + x];
        int x_begin = curCenterPos.x - d_rawImageParameter.m_xPixelBeginOffset - d_meanDispLenRadius;
        int y_begin = curCenterPos.y - d_rawImageParameter.m_yPixelBeginOffset - d_meanDispLenRadius;

        // 计算区域的宽度和高度
        int rectWidth = d_meanDispLenRadius * 2 + 1;
        int rectHeight = d_meanDispLenRadius * 2 + 1;

        // 计算该区域的均值
        float sum = 0.0f;
        int count = 0;

        for (int dy = 0; dy < rectHeight; dy++) {
            for (int dx = 0; dx < rectWidth; dx++) {
                int globalX = x_begin + dx;
                int globalY = y_begin + dy;

                // 确保访问的坐标在有效范围内
                if (globalX >= 0 && globalX < d_rawImageParameter.m_recImgWidth &&
                    globalY >= 0 && globalY < d_rawImageParameter.m_recImgHeight)
                {
                    sum += d_rawDisp[globalY * d_rawImageParameter.m_recImgWidth + globalX];
                    count++;
                }
            }
        }

        float meanDisp = sum / count;
        d_ppLensMeanDisp[y * d_rawImageParameter.m_xLensNum + x] = fmax(meanDisp, (float)d_disparityParameter.m_dispMin);
         
    }
}


void ImageRander::imageRanderWithOutMask(const DataParameter &dataParameter)
{
    RawImageParameter rawImageParameter = dataParameter.getRawImageParameter();
    MicroImageParameter microImageParameter = dataParameter.getMicroImageParameter();
    DisparityParameter disparityParameter = dataParameter.getDisparityParameter();

    // Define block and grid sizes
    dim3 blockSize(32, 32);
    dim3 gridSize((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, 
                  (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);

    computeLensMeanDispKernel<<<gridSize, blockSize>>>(d_rawDisp);

    // Check for any errors during kernel launch
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

	//cv::Mat randerMapinput, randerSceneMap, finalRanderMap;
	//dataParameter.m_inputImgRec.convertTo(randerMapinput, CV_64FC3);
	imageRander(rawImageParameter, microImageParameter,d_inputImgRec,3);
    saveThreeChannelGpuMemoryAsImage(d_randerMap,  h_randerMapHeight,  h_randerMapWidth, "result_3.bmp");
    imageRander(rawImageParameter, microImageParameter,d_rawDisp,1);
    saveSingleChannelGpuMemoryAsImage(d_randerMap, h_randerMapHeight,  h_randerMapWidth, "result_1.bmp");
    //imageRander_1(rawImageParameter, microImageParameter);
	//std::string storeName = dataParameter.m_folderPath + "/randerSceneMap.bmp";
	//randerSceneMap.convertTo(finalRanderMap, CV_8UC3);
	//imwrite(storeName, finalRanderMap);
}


__global__ void accumulateKernel(RanderMapPatch* d_ppRanderMapPatch,float* d_randerMap,float* d_randerCount,
    int DEST_WIDTH_, 
    int DEST_HEIGHT_, 
    int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // 获取 randerMapWidth 和 randerMapHeight 通过解引用指针
    int randerMapWidthVal = d_randerMapWidth[0];
    int randerMapHeightVal = d_randerMapHeight[0];

    if (x < d_rawImageParameter.m_xLensNum && y < d_rawImageParameter.m_yLensNum)
    {
        // 使用线性索引来访问 RanderMapPatch
        RanderMapPatch patch = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x];

        // 计算在输出图像中的起始点
        int sy_b = patch.sy - DEST_HEIGHT_ / 2;
        int sx_b = patch.sx - DEST_WIDTH_ / 2;

        // 处理补丁
        for (int py = 0; py < DEST_HEIGHT_; ++py)
        {
            for (int px = 0; px < DEST_WIDTH_; ++px)
            {
                int rander_x = sx_b + px;
                int rander_y = sy_b + py;

                if (rander_x >= 0 && rander_x < randerMapWidthVal && rander_y >= 0 && rander_y < randerMapHeightVal)
                {
                    // 将补丁添加到渲染图
                    for (int c = 0; c < channels; ++c)
                    {
                        atomicAdd(&d_randerMap[(rander_y * randerMapWidthVal + rander_x) * channels + c], patch.simg[(py * DEST_WIDTH_ + px) * channels + c]);
                    }

                    // 统计每个像素被多少个补丁贡献
                    atomicAdd(&d_randerCount[rander_y * randerMapWidthVal + rander_x], 1.0f);
                }
            }
        }
    }
}


__global__ void normalizeKernel(float* d_randerMap,float* d_randerCount,int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // 获取宽度和高度，通过解引用指针
    int randerMapWidth = d_randerMapWidth[0];
    int randerMapHeight = d_randerMapHeight[0];

    if (x < randerMapWidth && y < randerMapHeight)
    {
        for (int c = 0; c < channels; ++c)
        {
            int idx = (y * randerMapWidth + x) * channels + c;
            if (d_randerCount[y * randerMapWidth + x] > 0)
            {
                d_randerMap[idx] /= d_randerCount[y * randerMapWidth + x];
            }
        }
    }
}


__global__ void computeBoundaryKernel(RanderMapPatch* d_ppRanderMapPatch,
    int DEST_WIDTH_, int DEST_HEIGHT_)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    if (x < d_rawImageParameter.m_xLensNum && y < d_rawImageParameter.m_yLensNum &&
        x >= 0 && y >= 0) // Ensure x and y are within valid bounds
    {
        // 使用一维数组访问
        int sy = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sy;
        int sx = d_ppRanderMapPatch[y * d_rawImageParameter.m_xLensNum + x].sx;

        atomicMin(d_sx_begin, sx - DEST_WIDTH_ / 2);
        atomicMin(d_sy_begin, sy - DEST_HEIGHT_ / 2);
        atomicMax(d_sx_end, sx + DEST_WIDTH_ / 2);
        atomicMax(d_sy_end, sy + DEST_HEIGHT_ / 2);
    }
}


__global__ void processPatchKernel(RanderMapPatch* d_ppRanderMapPatch, float* d_input,
    int patchWidth, int patchHeight,int Channels) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = threadIdx.z;  // 处理的颜色通道 (0, 1, 2)

    // Adjust the range to match the CPU code
    int xAdjusted = x + d_rawImageParameter.m_xCenterBeginOffset;
    int yAdjusted = y + d_rawImageParameter.m_yCenterBeginOffset;

    if (xAdjusted >= d_rawImageParameter.m_xLensNum - d_rawImageParameter.m_xCenterEndOffset ||
        yAdjusted >= d_rawImageParameter.m_yLensNum - d_rawImageParameter.m_yCenterEndOffset) {
        return;
    }

    Point2d curCenterPos = d_microImageParameter.m_ppLensCenterPoints[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted];
    int blockSize = fabsf(roundf(d_ppLensMeanDisp[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted]));
    int starty = max(static_cast<int>(curCenterPos.y - blockSize / 2 - d_rawImageParameter.m_yPixelBeginOffset), 0);
    int startx = max(static_cast<int>(curCenterPos.x - blockSize / 2 - d_rawImageParameter.m_xPixelBeginOffset), 0);


    float *d_srcImg = d_input + (starty * d_rawImageParameter.m_xLensNum + startx) * Channels;
    float *d_simg = d_ppRanderMapPatch[yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted].simg;
    //printf("d_srcImg: %f\n", d_srcImg[0]);
    //printf("d_simg: %f\n", d_simg[0]);

    //printf("%d\n",d_simg[0]);
    // 计算当前线程处理的 Patch 位置
    int i = threadIdx.x;
    int j = threadIdx.y;

if (i < patchWidth && j < patchHeight) {
    // 计算双线性插值
    float fx = (float)i / (patchWidth - 1) * (blockSize - 1);
    float fy = (float)j / (patchHeight - 1) * (blockSize - 1);
    int ix = (int)fx;
    int iy = (int)fy;
    float wx = fx - ix;
    float wy = fy - iy;

    float top_left = d_srcImg[(iy * blockSize + ix) * Channels + c];
    float top_right = d_srcImg[(iy * blockSize + ix + 1) * Channels + c];
    float bottom_left = d_srcImg[((iy + 1) * blockSize + ix) * Channels + c];
    float bottom_right = d_srcImg[((iy + 1) * blockSize + ix + 1) * Channels + c];

    float interpolated = (1 - wx) * (1 - wy) * top_left +
                        wx * (1 - wy) * top_right +
                        (1 - wx) * wy * bottom_left +
                        wx * wy * bottom_right;

    //printf("x: %d, y: %d, i: %d, j: %d, interpolated: %f\n", xAdjusted, yAdjusted, i, j, interpolated);
    //printf("x: %d, y: %d, i: %d, j: %d, interpolated: %f\n", xAdjusted, yAdjusted, i, j, d_simg[(j * patchWidth + i) * 3 + c]);
    d_simg[(j * patchWidth + i) * Channels + c] = interpolated;

    // 镜像翻转 Patch
    d_simg[(j * patchWidth + (patchWidth - 1 - i)) * Channels + c] = interpolated;

    // 存储 Patch 位置
        if (c == 0 && i == 0 && j == 0) {
            int patchIdx = yAdjusted * d_rawImageParameter.m_xLensNum + xAdjusted;
            d_ppRanderMapPatch[patchIdx].sy = curCenterPos.y;
            d_ppRanderMapPatch[patchIdx].sx = curCenterPos.x;
        }
    }
}




void ImageRander::imageRander(const RawImageParameter &rawImageParameter, 
    const MicroImageParameter &microImageParameter,float* d_input,int Channels)
{
   
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Step 1: Process patch kernel
    hipEventRecord(start); 
    dim3 blockSize(16, 16, Channels);  // (Patch 16x16, 每个线程负责一个像素，3 个通道)
    dim3 gridSize((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, 
                  (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
    
    processPatchKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,d_input,DEST_WIDTH, DEST_HEIGHT,Channels);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Process Patch Kernel Time: %f ms\n", ms); 



    // Step 2: Compute boundary kernel
    hipEventRecord(start); 
    blockSize = dim3(32, 32);
    gridSize = dim3((rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x, (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y);
    computeBoundaryKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,DEST_WIDTH, DEST_HEIGHT);
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Compute Boundary Kernel Time: %f ms\n", ms); 


    // Step 3: Compute width and height kernel

    int h_sx_begin, h_sy_begin, h_sx_end, h_sy_end;
    CUDA_CHECK(hipMemcpyFromSymbol(&h_sx_begin, HIP_SYMBOL(d_sx_begin), sizeof(int)));
    CUDA_CHECK(hipMemcpyFromSymbol(&h_sy_begin, HIP_SYMBOL(d_sy_begin), sizeof(int)));
    CUDA_CHECK(hipMemcpyFromSymbol(&h_sx_end, HIP_SYMBOL(d_sx_end), sizeof(int)));
    CUDA_CHECK(hipMemcpyFromSymbol(&h_sy_end, HIP_SYMBOL(d_sy_end), sizeof(int)));

    h_randerMapWidth = h_sx_end - h_sx_begin + 1;
    h_randerMapHeight = h_sy_end - h_sy_begin + 1;

    size_t randerMapSize = h_randerMapWidth * h_randerMapHeight * Channels * sizeof(float); 
    size_t randerCountSize = h_randerMapWidth * h_randerMapHeight * sizeof(float);


    CUDA_CHECK(hipMalloc(&d_randerMap, randerMapSize));
    CUDA_CHECK(hipMalloc(&d_randerCount, randerCountSize));

    // 初始化设备内存
    CUDA_CHECK(hipMemset(d_randerMap, 0, randerMapSize));
    CUDA_CHECK(hipMemset(d_randerCount, 0, randerCountSize));

    // Step 5: Accumulate kernel
    hipEventRecord(start); // 记录开始时间
    gridSize.x = (rawImageParameter.m_xLensNum + blockSize.x - 1) / blockSize.x;
    gridSize.y = (rawImageParameter.m_yLensNum + blockSize.y - 1) / blockSize.y;

    accumulateKernel<<<gridSize, blockSize>>>(d_ppRanderMapPatch,d_randerMap,d_randerCount,DEST_WIDTH, DEST_HEIGHT, Channels); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Accumulate Kernel Time: %f ms\n", ms); 


    // Step 6: Normalize kernel
    hipEventRecord(start); 
    gridSize.x = (h_randerMapWidth + blockSize.x - 1) / blockSize.x;
    gridSize.y = (h_randerMapHeight + blockSize.y - 1) / blockSize.y;
    normalizeKernel<<<gridSize, blockSize>>>(d_randerMap,d_randerCount,Channels); // 3通道
    CUDA_CHECK(hipGetLastError()); 
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(stop); 
    hipEventSynchronize(stop); 
    hipEventElapsedTime(&ms, start, stop);
    printf("Normalize Kernel Time: %f ms\n", ms); 


   
    hipEventDestroy(start);
    hipEventDestroy(stop);
}




/*__global__ void resizeKernel(float *d_src, float *d_dst, int src_width, int src_height, int dst_width, int dst_height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dst_width && y < dst_height)
    {
        // 计算源图像对应位置的坐标
        float fx = (float)x / (dst_width - 1) * (src_width - 1);
        float fy = (float)y / (dst_height - 1) * (src_height - 1);
        
        int ix = (int)fx;
        int iy = (int)fy;

        // 获取邻域的像素值并执行双线性插值
        float wx = fx - ix;
        float wy = fy - iy;

        for (int c = 0; c < channels; ++c)
        {
            float top_left = d_src[(iy * src_width + ix) * channels + c];
            float top_right = d_src[(iy * src_width + (ix + 1)) * channels + c];
            float bottom_left = d_src[((iy + 1) * src_width + ix) * channels + c];
            float bottom_right = d_src[((iy + 1) * src_width + (ix + 1)) * channels + c];

            float interpolated = (1 - wx) * (1 - wy) * top_left +
                                 wx * (1 - wy) * top_right +
                                 (1 - wx) * wy * bottom_left +
                                 wx * wy * bottom_right;

            d_dst[(y * dst_width + x) * channels + c] = interpolated;
        }
    }
}

void resizeCUDA(float *d_src, float *d_dst, int src_width, int src_height, int dst_width, int dst_height, int channels)
{
    dim3 blockSize(16, 16);
    dim3 gridSize((dst_width + blockSize.x - 1) / blockSize.x, (dst_height + blockSize.y - 1) / blockSize.y);

    resizeKernel<<<gridSize, blockSize>>>(d_src, d_dst, src_width, src_height, dst_width, dst_height, channels);
    hipDeviceSynchronize();
}

__global__ void flipKernel(float *d_src, float *d_dst, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // 水平翻转：交换左右位置
        int flip_x = width - 1 - x;

        for (int c = 0; c < channels; ++c)
        {
            d_dst[(y * width + x) * channels + c] = d_src[(y * width + flip_x) * channels + c];
        }
    }
}

void flipCUDA(float *d_src, float *d_dst, int width, int height, int channels)
{
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    flipKernel<<<gridSize, blockSize>>>(d_src, d_dst, width, height, channels);
    hipDeviceSynchronize();
}






void ImageRander::imageRander(float **ppLensMeanDisp, const RawImageParameter &rawImageParameter, const MicroImageParameter &microImageParameter, cv::Mat &randerImg, cv::Mat &destImg)
{
    RanderMapPatch **ppRanderMapPatch = new RanderMapPatch *[rawImageParameter.m_yLensNum];
    for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
        ppRanderMapPatch[i] = new RanderMapPatch[rawImageParameter.m_xLensNum];

    float *d_randerImg, *d_tmp, *d_simg;
    size_t imgSize = randerImg.rows * randerImg.cols * randerImg.channels() * sizeof(float);
    hipMalloc(&d_randerImg, imgSize);
    hipMemcpy(d_randerImg, randerImg.ptr<float>(), imgSize, hipMemcpyHostToDevice);

    size_t tmpSize = DEST_WIDTH * DEST_HEIGHT * randerImg.channels() * sizeof(float);
    hipMalloc(&d_tmp, tmpSize);
    hipMalloc(&d_simg, tmpSize);

    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            int blockSize = fabs(std::round(d_ppLensMeanDisp[y * rawImageParameter.m_xLensNum + x]));

            Point2d &curCenterPos = microImageParameter.m_ppLensCenterPoints[y][x];
            int starty = curCenterPos.y - blockSize / 2 - rawImageParameter.m_yPixelBeginOffset;
            int startx = curCenterPos.x - blockSize / 2 - rawImageParameter.m_xPixelBeginOffset;

            float *d_srcImg = d_randerImg + (starty * randerImg.cols + startx) * randerImg.channels();
            resizeCUDA(d_srcImg, d_tmp, blockSize, blockSize, DEST_WIDTH, DEST_HEIGHT, randerImg.channels());
            flipCUDA(d_tmp, d_simg, DEST_WIDTH, DEST_HEIGHT, randerImg.channels());

            ppRanderMapPatch[y][x].sy = curCenterPos.y;
            ppRanderMapPatch[y][x].sx = curCenterPos.x;
            ppRanderMapPatch[y][x].simg = d_simg;
        }
    }

    int sx_begin = INT_MAX, sy_begin = INT_MAX;
    int sx_end = INT_MIN, sy_end = INT_MIN;

    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            sy_begin = std::min(sy_begin, ppRanderMapPatch[y][x].sy - DEST_HEIGHT / 2);
            sx_begin = std::min(sx_begin, ppRanderMapPatch[y][x].sx - DEST_WIDTH / 2);
            sy_end = std::max(sy_end, ppRanderMapPatch[y][x].sy + DEST_HEIGHT / 2);
            sx_end = std::max(sx_end, ppRanderMapPatch[y][x].sx + DEST_WIDTH / 2);
        }            
    }

    int randerMapWidth = sx_end - sx_begin + 1;
    int randerMapHeight = sy_end - sy_begin + 1;

    float *d_randerMap, *d_randerCount;
    size_t randerMapSize = randerMapWidth * randerMapHeight * randerImg.channels() * sizeof(float);
    size_t randerCountSize = randerMapWidth * randerMapHeight * sizeof(float);
    hipMalloc(&d_randerMap, randerMapSize);
    hipMalloc(&d_randerCount, randerCountSize);
    hipMemset(d_randerMap, 0, randerMapSize);
    hipMemset(d_randerCount, 0, randerCountSize);

    dim3 blockSize(16, 16);
    for (int y = rawImageParameter.m_yCenterBeginOffset; y < rawImageParameter.m_yLensNum - rawImageParameter.m_yCenterEndOffset; y++)
    {
        for (int x = rawImageParameter.m_xCenterBeginOffset; x < rawImageParameter.m_xLensNum - rawImageParameter.m_xCenterEndOffset; x++)
        {
            int sy_b = ppRanderMapPatch[y][x].sy - DEST_HEIGHT / 2 - sy_begin;
            int sx_b = ppRanderMapPatch[y][x].sx - DEST_WIDTH / 2 - sx_begin;

            dim3 gridSize((DEST_WIDTH + blockSize.x - 1) / blockSize.x, (DEST_HEIGHT + blockSize.y - 1) / blockSize.y);
            accumulateKernel<<<gridSize, blockSize>>>(d_randerMap, d_randerCount, ppRanderMapPatch[y][x].simg, sx_b, sy_b, DEST_WIDTH, DEST_HEIGHT, randerMapWidth, randerMapHeight, randerImg.channels());
        }
    }

    dim3 gridSize((randerMapWidth + blockSize.x - 1) / blockSize.x, (randerMapHeight + blockSize.y - 1) / blockSize.y);
    normalizeKernel<<<gridSize, blockSize>>>(d_randerMap, d_randerCount, randerMapWidth, randerMapHeight, randerImg.channels());

    float *h_randerMap = new float[randerMapWidth * randerMapHeight * randerImg.channels()];
    hipMemcpy(h_randerMap, d_randerMap, randerMapSize, hipMemcpyDeviceToHost);

    destImg = cv::Mat(randerMapHeight, randerMapWidth, randerImg.type(), h_randerMap);

    hipFree(d_randerMap);
    hipFree(d_randerCount);
    hipFree(d_randerImg);
    hipFree(d_tmp);
    hipFree(d_simg);

    for (int i = 0; i < rawImageParameter.m_yLensNum; i++)
        delete[] ppRanderMapPatch[i];
    delete[] ppRanderMapPatch;
}*/

void ImageRander::outputSparseSceneDepth(string folderName, cv::Mat &sceneSparseDepth, cv::Mat &sceneDepthMask)
{
	std::string storeName;
	ofstream ofs1;
	storeName = folderName + "/sceneInitDisp.txt";
	ofs1.open(storeName, ofstream::out);
	ofstream ofs2;
	storeName = folderName + "/sceneInitDispMask.txt";
	ofs2.open(storeName, ofstream::out);

	for (int y = 0; y < sceneSparseDepth.rows; ++y)
	{
		double *pYSceneDisp = (double *)sceneSparseDepth.ptr<double>(y);
		uchar *pYMask = (uchar *)sceneDepthMask.ptr<uchar>(y);
		for (int x = 0; x < sceneSparseDepth.cols; ++x)
		{
			ofs1 << fixed << setprecision(5) << pYSceneDisp[x] << " ";
			if (pYMask[x] == 255)
				ofs2 << 1 << " ";
			else
				ofs2 << 0 << " ";
			//ofs2 << int(pYMask[x]) << " ";
		}
		ofs1 << endl;
		ofs2 << endl;
	}
	ofs1.close();
	ofs2.close();
}